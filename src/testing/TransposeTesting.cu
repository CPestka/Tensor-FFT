#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include "../base/Transposer.cu"


int main() {
  int fft_length = 16*16*16*16*16 * 16*2;
  // __half2* dptr_data;
  // __half2* dptr_results;
  //
  // hipMalloc(&dptr_data, 2 * sizeof(__half2) * fft_length);
  // dptr_results = dptr_data + fft_length;
  //
  // TransposeKernel<<<fft_length/512, 512>>>(
  //     dptr_data, dptr_results, fft_length, 5, 1);

  __half* dptr_data_RE;
  __half* dptr_results_RE;
  __half* dptr_data_IM;
  __half* dptr_results_IM;

  hipMalloc(&dptr_data_RE, 4 * sizeof(__half) * fft_length);
  dptr_data_IM = dptr_data_RE + fft_length;
  dptr_results_RE = dptr_data_IM + fft_length;
  dptr_results_IM = dptr_results_RE + fft_length;

  TransposeKernel<<<fft_length/512, 512>>>(
      dptr_data_RE, dptr_data_IM, dptr_results_RE, dptr_results_IM,  fft_length, 5, 1);
}
