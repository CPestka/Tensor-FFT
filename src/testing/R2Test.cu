#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <memory>

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "TestingDataCreation.h"
#include "FileWriter.h"
#include "../base/Radix2.cu"

int main(){
  constexpr int fft_length = 4;
  std::vector<float> weights_RE { 1.0 };
  std::vector<float> weights_IM { 0.0 };
  std::unique_ptr<__half[]> data =
      CreateSineSuperpostion(fft_length, weights_RE, weights_IM);

  WriteResultsToFile("r2_in.dat", fft_length, data.get());

  std::unique_ptr<__half[]> preped_data =
      std::make_unique<__half[]>(2 * fft_length);

  for(int i=0; i<fft_length/2; i++){
    preped_data[i] = data[2 * i];
    preped_data[i + (fft_length / 2)] = data[(2 * i) + 1];
  }
  WriteResultsToFile("r2_preped.dat", fft_length, preped_data.get());

  __half* dptr_data;
  hipMalloc(&dptr_data, sizeof(__half) * 4 * fft_length);

  hipMemcpy(dptr_data, preped_data.get(),
             sizeof(__half) * 2 * fft_length, hipMemcpyHostToDevice);

  __half* data_RE = dptr_data;
  __half* data_IM = dptr_data + fft_length;
  __half* results_RE = dptr_data + (2 * fft_length);
  __half* results_IM = dptr_data + (3 * fft_length);

  Radix2Kernel<<<1, fft_length / 4>>>(
      data_RE, data_IM, results_RE, results_IM, 1);
  Radix2Kernel<<<1, fft_length / 4>>>(
      data_RE + 2, data_IM + 2, results_RE + 2, results_IM + 2, 1);

  std::swap(data_RE, results_RE);
  std::swap(data_IM, results_IM);

  Radix2Kernel<<<1, fft_length / 2>>>(
      data_RE, data_IM, results_RE, results_IM, 2);


  hipMemcpy(data.get(), results_RE,
             sizeof(__half) * 2 * fft_length, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  WriteResultsToFile("r2_out.dat", fft_length, data.get());
}
