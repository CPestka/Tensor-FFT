#include "hip/hip_runtime.h"
#pragma once
//Provides comparision for the fft test via cuFFT
#include <iostream>
#include <memory>
#include <vector>
#include <assert.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_fp16.h>

#include "../FileWriter.cu"
#include "../TestingDataCreation.cu"


bool cuFFT_16(){
  long long fft_length = 16*16;
  std::vector<float> weights;
  weights.push_back(1.0);
  std::unique_ptr<__half2[]> data =
      CreateSineSuperpostionH2(fft_length,  weights);
  weights.clear();
  weights.push_back(0.0);
  std::unique_ptr<__half2[]> results =
      CreateSineSuperpostionH2(fft_length,  weights);

  __half2* dptr_data;
  __half2* dptr_results;
  hipMalloc(&dptr_data, sizeof(__half2) * fft_length);
  hipMalloc(&dptr_results, sizeof(__half2) * fft_length);
  hipMemcpy(dptr_data, data.get(), fft_length * sizeof(__half2),
             hipMemcpyHostToDevice);

  WriteResultsToFileHalf2("test_fft_cuFFTinput.dat", fft_length, data.get());

  hipfftHandle plan;
  hipfftResult r;

  r = hipfftCreate(&plan);
  if (r != HIPFFT_SUCCESS) {
    return false;
  }

  size_t size = 0;
  r = hipfftXtMakePlanMany(plan, 1, &fft_length, nullptr, 1, 1, HIP_C_16F,
                          nullptr, 1, 1, HIP_C_16F, 1, &size, HIP_C_16F);
  if (r != HIPFFT_SUCCESS) {
    return false;
  }

  r = hipfftXtExec(plan, dptr_data, dptr_results, HIPFFT_FORWARD);
  if (r != HIPFFT_SUCCESS) {
    return false;
  }

  hipMemcpy(results.get(), dptr_results, fft_length * sizeof(__half2),
             hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  WriteResultsToFileHalf2("test_fft_cuFFTresults.dat", fft_length,
                          results.get());

  hipfftDestroy(plan);
  hipFree(dptr_results);
  hipFree(dptr_data);

  return true;
}

bool cuFFT_2(){
  long long fft_length = 16*16*16*16*16*2*2*2;
  std::vector<float> weights;
  weights.push_back(1.0);
  std::unique_ptr<__half2[]> data =
      CreateSineSuperpostionH2(fft_length,  weights);
  weights.clear();
  weights.push_back(0.0);
  std::unique_ptr<__half2[]> results =
      CreateSineSuperpostionH2(fft_length,  weights);

  __half2* dptr_data;
  __half2* dptr_results;
  hipMalloc(&dptr_data, sizeof(__half2) * fft_length);
  hipMalloc(&dptr_results, sizeof(__half2) * fft_length);
  hipMemcpy(dptr_data, data.get(), fft_length * sizeof(__half2),
             hipMemcpyHostToDevice);

  WriteResultsToFileHalf2("test_fft_cuFFTinput.dat", fft_length, data.get());

  hipfftHandle plan;
  hipfftResult r;

  r = hipfftCreate(&plan);
  if (r != HIPFFT_SUCCESS) {
    return false;
  }

  size_t size = 0;
  r = hipfftXtMakePlanMany(plan, 1, &fft_length, nullptr, 1, 1, HIP_C_16F,
                          nullptr, 1, 1, HIP_C_16F, 1, &size, HIP_C_16F);
  if (r != HIPFFT_SUCCESS) {
    return false;
  }

  r = hipfftXtExec(plan, dptr_data, dptr_results, HIPFFT_FORWARD);
  if (r != HIPFFT_SUCCESS) {
    return false;
  }

  hipMemcpy(results.get(), dptr_results, fft_length * sizeof(__half2),
             hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  WriteResultsToFileHalf2("test_fft_cuFFTresults.dat", fft_length,
                          results.get());

  hipfftDestroy(plan);
  hipFree(dptr_results);
  hipFree(dptr_data);

  return true;
}
