#include "hip/hip_runtime.h"
//Used to test functonality
#include <iostream>
#include <string>

#include "TransposeTest.cu"
#include "DFTMatrixTest.cu"
#include "DFTTest.cu"
#include "FFTTest.cu"
#include "CuFFTTest.cu"

int main(){
  if (!transpose16_test()){
    std::cout << "Transpose 16*16*16 test failed!" << std::endl;
    return false;
  }
  if (!transpose16_2_test()){
    std::cout << "Transpose 16*16*16*2*2 test failed!" << std::endl;
    return false;
  }

  if (!dft_matrix16_test()){
    std::cout << "DFT matrix test 16*16 failed!" << std::endl;
    return false;
  }
  if (!dft_matrix16_2_test()){
    std::cout << "DFT matrix test 16*16*16*16*2 failed!" << std::endl;
    return false;
  }

  if (!dft_0_test()){
    std::cout << "DFT kernel test failed for 16*16*16 with value 0!"
              << std::endl;
    return false;
  }
  if (!dft_sin_test_16()){
    std::cout << "DFT kernel test failed for 16*16*16 with sin(x) x [0:2*PI]!"
              << std::endl;
    return false;
  }
  if (!dft_sin_test_2()){
    std::cout << "DFT kernel test failed for 16*16*16*16*2 with sin(x) x"
              << " [0:2*PI]!"
              << std::endl;
    return false;
  }

  if (!full_test(16*16, "test_fft_16_2")){
    std::cout << "FFT test for a length of 16*16 failed." << std::endl;
    return false;
  }
  if (!full_test(16*16*16, "test_fft_16_3")){
    std::cout << "FFT test for a length of 16*16*16 failed." << std::endl;
    return false;
  }
  if (!full_test(16*16*16*2*2*2, "test_fft_16_3_2_3")){
    std::cout << "FFT test for a length of 16^3*2^3 failed." << std::endl;
    return false;
  }
  /*
  if (!full_test(16*16*16*16*16*2*2*2, "test_fft_16_5_2_3")){
    std::cout << "FFT test for a length of 16^5*2^3 failed." << std::endl;
    return false;
  }

  if (!compute_fft_cuFFT(16*16)){
    std::cout << "Generation of comparision data for FFT 16*16 test by cuFFT"
              << " failed." << std::endl;
    return false;
  }
  if (!compute_fft_cuFFT(16*16*16)){
    std::cout << "Generation of comparision data for FFT 16^3 test by cuFFT"
              << " failed." << std::endl;
    return false;
  }
  if (!compute_fft_cuFFT(16*16*16*2*2*2)){
    std::cout << "Generation of comparision data for FFT 16^3*2^3 test by cuFFT"
              << " failed." << std::endl;
    return false;
  }
  if (!compute_fft_cuFFT(16*16*16*16*16*2*2*2)){
    std::cout << "Generation of comparision data for FFT 16^5*2^3 test by cuFFT"
              << " failed." << std::endl;
    return false;
  }
  */

  std::cout << "All tests passed! Results of full FFT test have to be check"
            << " manually" <<std::endl;

  return true;
}
