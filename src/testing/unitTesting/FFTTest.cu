#include "hip/hip_runtime.h"
#pragma once

//Used to test functonality
#include <iostream>
#include <string>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "../../base/Plan.cpp"
#include "../../base/ComputeFFT.cu"
#include "../TestingDataCreation.cu"
#include "../FileWriter.cu"
#include "../Timer.h"

bool full_test(int fft_length, std::string file_name_prefix){
  IntervallTimer my_timer();

  //Prepare input data on cpu
  std::vector<float> weights;
  weights.push_back(1.0);
  std::unique_ptr<__half[]> data = CreateSineSuperpostion(fft_length, weights);

  //Get plan
  Plan my_plan;
  if (CreatePlan(fft_length)) {
    my_plan = CreatePlan(fft_length).value();
  } else {
    std::cout << "Plan creation failed" << std::endl;
    return false;
  }

  WriteResultsREToFile(file_name_prefix + "_input.dat", fft_length, data.get());

  std::string error_mess;

  //Construct a DataHandler for data on GPU
  DataHandler my_handler(fft_length);
  error_mess = my_handler.PeakAtLastError().value_or("");
  if (error_mess != "") {
    std::cout << error_mess << std::endl;
    return false;
  }

  //Copy data to gpu
  error_mess = my_handler.CopyDataHostToDevice(data.get()).value_or("");
  if (error_mess != "") {
    std::cout << error_mess << std::endl;
    return false;
  }

  //Compute FFT
  error_mess = ComputeFFT(my_plan, my_handler).value_or("");
  if (error_mess != "") {
    std::cout << error_mess << std::endl;
    return false;
  }

  //Copy results back to cpu
  error_mess = my_handler.CopyResultsDeviceToHost(
      data.get(), my_plan.amount_of_r16_steps_,
      my_plan.amount_of_r2_steps_).value_or("");
  if (error_mess != "") {
    std::cout << error_mess << std::endl;
    return false;
  }

  hipDeviceSynchronize();

  WriteResultsToFile(file_name_prefix + "_results.dat", fft_length, data.get());

  my_timer.getTimeInMilliseconds();

  return true;
}
