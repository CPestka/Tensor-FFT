#include "hip/hip_runtime.h"
//Used to profile the function ComputeFFT
#include <iostream>
#include <vector>
#include <optional>
#include <string>
#include <memory>

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "TestingDataCreation.h"

int main(){
  long long fft_length = 16*16*16;
  constexpr int batch_size = 20;

  std::vector<float> weights;
  weights.push_back(1.0);
  std::unique_ptr<__half2[]> data =
      CreateSineSuperpostionH2Batch(fft_length,  weights, batch_size);

  __half2* dptr_data;
  __half2* dptr_results;
  hipMalloc(&dptr_data, sizeof(__half2) * fft_length * batch_size);
  hipMalloc(&dptr_results, sizeof(__half2) * fft_length * batch_size);

  hipfftHandle plan;
  hipfftResult r;

  r = hipfftCreate(&plan);
  if (r != HIPFFT_SUCCESS) {
    std::cout << "Error! Plan creation failed." << std::endl;
    return false;
  }

  size_t size = 0;
  r = hipfftXtMakePlanMany(plan, 1, &fft_length, nullptr, 1, 1,
                          HIP_C_16F, nullptr, 1, 1, HIP_C_16F, batch_size,
                          &size, HIP_C_16F);
  if (r != HIPFFT_SUCCESS) {
    std::cout << "Error! Plan creation failed." << std::endl;
    return false;
  }

  hipMemcpy(dptr_data, data.get(), fft_length * sizeof(__half2) * batch_size,
             hipMemcpyHostToDevice);

  r = hipfftXtExec(plan, dptr_data, dptr_results, HIPFFT_FORWARD);
  if (r != HIPFFT_SUCCESS) {
    std::cout << "Error! Plan execution failed." << std::endl;
    return false;

  hipfftDestroy(plan);
  hipFree(dptr_results);
  hipFree(dptr_data);

  return true;
}
