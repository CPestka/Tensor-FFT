#include "hip/hip_runtime.h"
//
#include <iostream>
#include <memory>
#include <vector>
#include <assert.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_fp16.h>

#include "../base/Timer.h"
#include "FileWriter.cu"
#include "TestingDataCreation.cu"


int main(){
  long long fft_length = 16*16*16;
  std::vector<float> weights;
  weights.push_back(1.0);
  std::unique_ptr<__half2[]> data =
      CreateSineSuperpostionH2(fft_length,  weights);
  weights.clear();
  weights.push_back(0.0);
  std::unique_ptr<__half2[]> results =
      CreateSineSuperpostionH2(fft_length,  weights);

  __half2* dptr_data;
  __half2* dptr_results;
  hipMalloc(&dptr_data, sizeof(__half2) * fft_length);
  hipMalloc(&dptr_results, sizeof(__half2) * fft_length);
  hipMemcpy(dptr_data, data.get(), fft_length * sizeof(__half2),
             hipMemcpyHostToDevice);

  WriteResultsToFileHalf2("cuFFTinput.dat", fft_length, data.get());

  hipfftHandle plan;
  hipfftResult r;
  r = hipfftCreate(&plan);
  assert(r == HIPFFT_SUCCESS);
  size_t size = 0;
  r = hipfftXtMakePlanMany(plan, 1, &fft_length, nullptr, 1, 1, HIP_C_16F,
                          nullptr, 1, 1, HIP_C_16F, 1, &size, HIP_C_16F);
  assert(r == HIPFFT_SUCCESS);
  IntervallTimer timer;
  r = hipfftXtExec(plan, dptr_data, dptr_results, HIPFFT_FORWARD);
  assert(r == HIPFFT_SUCCESS);
  std::cout << "Computation took: " << timer.getTimeInMilliseconds() << " ms"
            << std::endl;

  hipMemcpy(results.get(), dptr_results, fft_length * sizeof(__half2),
             hipMemcpyDeviceToHost);
  WriteResultsToFileHalf2("cuFFTresults.dat", fft_length, results.get());

  hipFree(dptr_data);
  hipFree(dptr_results);
}
