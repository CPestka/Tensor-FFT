#include "hip/hip_runtime.h"
#include <iostream>
#include <memory>
#include <fstream>
#include <string>
#include <iomanip>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

int main() {
  constexpr int n = 16;
  constexpr int m = 16;

  std::unique_ptr<__half[]> twiddle = std::make_unique<__half[]>(2*n*m);

  for(int i=0; i<n; i++){
    for(int j=0; j<m; j++){
      double phase = (2 * M_PI * j * i) / (n * m);
      double re = cos(phase);
      double im = -sin(phase);
      twiddle[j + (m * i)] = static_cast<__half>(re);
      twiddle[j + (m * i) + (m * n)] = static_cast<__half>(im);
    }
  }

  std::string file_name =
      (((("twiddle_" + std::to_string(n)) + "_") + std::to_string(m)) + ".dat");
  std::ofstream myfile (file_name);
  if (myfile.is_open()) {
    file_name << std::setprecision(20);
    for(int j=0; j<n; j++){
      for(int i=0; i<m; i++){
        file_name << static_cast<double>(twiddle[i + (j * m)]) << ", ";
      }
      file_name << "/n";
    }

    file_name << "/n";
    for(int j=0; j<n; j++){
      for(int i=0; i<m; i++){
        file_name << static_cast<double>(twiddle[i + (j * m) + (m * n)]) << ", ";
      }
      file_name << "/n";
    }

    myfile.close();
  }
}
