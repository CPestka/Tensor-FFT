#include "hip/hip_runtime.h"
#include <iostream>
#include <memory>

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void Twiddle(__half* output, int length_halfed){
  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;

  for(int k=0; k<16; k++){
    output[thread_id] =
        static_cast<__half>(cos((M_PI * thread_id * k)/length_halfed));
    output[thread_id + 32 * length_halfed] =
        static_cast<__half>(-sin((M_PI * thread_id * k)/length_halfed));

    output[thread_id + 64 * length_halfed] =
        static_cast<__half>(cosf((M_PI * thread_id * k)/length_halfed));
    output[thread_id + 64 * length_halfed + 32 * length_halfed] =
        static_cast<__half>(-sinf((M_PI * thread_id * k)/length_halfed));

    output[thread_id + 128 * length_halfed] =
        hcos(__hdiv(__hmul(static_cast<__half>(M_PI),
                           static_cast<__half>(thread_id * k)),
                    static_cast<__half>(length_halfed)));
    output[thread_id + 128 * length_halfed + 32 * length_halfed] =
        hcos(__hdiv(__hmul(static_cast<__half>(M_PI),
                           static_cast<__half>(thread_id * k)),
                    static_cast<__half>(length_halfed)));
  }

}

int main() {
  int n = 16;
  int m = 16;

  std::unique_ptr<__half[]> results = std::make_unique<__half[]>(2 * 3 * n * m);

  __half* dptr_results;
  hipMalloc((void**)(&dptr_results), 3 * sizeof(__half) * 2 * n * m);

  Twiddle<<<1, 16>>>(dptr_results, m/2);

  hipDeviceSynchronize();

  hipMemcpy(results.get(), dptr_results, 3 * sizeof(__half) * 2 * n * m,
             hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  for(int i=0; i<n*m; i++){
    double fp32_trig_RE = results[i];
    double fp32_trig_IM = results[i + (n*m)];

    double fast_trig_RE = results[i + (2*n*m)];
    double fast_trig_IM = results[i + (3*n*m)];

    double half_trig_RE = results[i + (4*n*m)];
    double half_trig_IM = results[i + (5*n*m)];

    if (fp32_trig_RE != fast_trig_RE) {
      std::cout << "1_RE" << std::endl;
    }
    if (fp32_trig_IM != fast_trig_IM) {
      std::cout << "1_IM" << std::endl;
    }

    if (fp32_trig_RE != half_trig_RE) {
      std::cout << "2_RE" << std::endl;
    }
    if (fp32_trig_IM != half_trig_IM) {
      std::cout << "2_IM" << std::endl;
    }
  }

  for(int i=0; i<n; i++){
    for(int j=0; j<m; j++){
      std::cout << results[j + (i * m)] << "\t";
    }
    std::cout << std::endl;
  }
}
