#include "hip/hip_runtime.h"
//Used to test correctness of the dft kernel

#pragma once

#include <iostream>
#include <cstdint>
#include <string>
#include <vector>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <assert.h>

#include "TestingDataCreation.cu"
#include "FileWriter.cu"
#include "../base/Transposer.cu"
#include "../base/TensorDFT16.cu"
#include "../base/ComputeDFTMatrix.cu"

__global__ void PrepareCuFFTInput(__half* input_RE, __half* input_IM,
                                  __half2* cuFFT_in){
  for(int i=0; i<16; i++){
    cuFFT_in[i] = __halves2half2(input_RE[i], input_IM[i]);
  }
}

__global__ void SaveCuFFTResults(__half2* cuFFT_out, __half* out_RE,
                                 __half* out_IM){
  for(int i=0; i<16; i++){
    out_RE[i] = __low2half(cuFFT_out[i]);
    out_IM[i] = __high2half(cuFFT_out[i]);
  }
}

bool dft16_test(){
  int fft_length = 16*16*16;

  std::vector<float> weights;
  weights.push_back(1.0);
  std::unique_ptr<__half[]> data_1 =
      CreateSineSuperpostion(fft_length, weights);
  std::unique_ptr<__half[]> data_2 =
      CreateSineSuperpostion(fft_length, weights);

  WriteResultsToFile("input.dat", fft_length, data_1.get());

  __half* dptr_input_RE;
  __half* dptr_input_IM;
  __half* dptr_results_kernel_RE;
  __half* dptr_results_kernel_IM;
  __half* dptr_results_cuFFT_RE;
  __half* dptr_results_cuFFT_IM;
  hipMalloc((void**)(&dptr_input_RE), 6 * sizeof(__half) * fft_length);

  dptr_input_IM = dptr_input_RE + fft_length;
  dptr_results_kernel_RE = dptr_input_IM + fft_length;
  dptr_results_kernel_IM = dptr_results_kernel_RE + fft_length;
  dptr_results_cuFFT_RE = dptr_results_kernel_IM + fft_length;
  dptr_results_cuFFT_IM = dptr_results_cuFFT_RE + fft_length;

  hipMemcpy(dptr_input_RE, data_1.get(), 2 * fft_length * sizeof(__half),
             hipMemcpyHostToDevice);

  int transpose_blocksize = 256;
  int amount_of_transpose_blocks =
     ceil(static_cast<float>(fft_length) /
          static_cast<float>(transpose_blocksize));

  TransposeKernel<<<amount_of_transpose_blocks, transpose_blocksize>>>(
      dptr_input_RE, dptr_input_IM, dptr_results_kernel_RE,
      dptr_results_kernel_IM, fft_length, 2, 0);

  __half2* dptr_cuFFT_in;
  __half2* dptr_cuFFT_out;
  hipMalloc((void**)(&dptr_cuFFT_in), sizeof(__half2) * 16);
  hipMalloc((void**)(&dptr_cuFFT_out), sizeof(__half2) * 16);

  hipfftHandle plan;
  hipfftResult r;
  r = hipfftCreate(&plan);
  assert(r == HIPFFT_SUCCESS);
  size_t size = 0;
  long long fft_length_1 = 16;
  r = hipfftXtMakePlanMany(plan, 1, &fft_length_1, nullptr, 1, 1, HIP_C_16F,
                          nullptr, 1, 1, HIP_C_16F, 1, &size, HIP_C_16F);
  assert(r == HIPFFT_SUCCESS);

  for(int i=0; i<16*16; i++){
    int offset = 16*i;
    PrepareCuFFTInput<<<1,1>>>(dptr_results_kernel_RE + offset,
                               dptr_results_kernel_IM + offset,
                               dptr_cuFFT_in);
    hipDeviceSynchronize();

    r = hipfftXtExec(plan, dptr_cuFFT_in, dptr_cuFFT_out, HIPFFT_FORWARD);
    assert(r == HIPFFT_SUCCESS);

    hipDeviceSynchronize();

    SaveCuFFTResults<<<1,1>>>(dptr_cuFFT_out, dptr_results_cuFFT_RE + offset,
                              dptr_results_cuFFT_IM + offset);

    hipDeviceSynchronize();

  }

  hipMemcpy(data_2.get(), dptr_results_cuFFT_RE,
             2 * fft_length * sizeof(__half), hipMemcpyDeviceToHost);

  WriteResultsToFile("dft_test_cuFFT.dat", fft_length, data_2.get());

  __half* dptr_dft_matrix_batch_RE;
  __half* dptr_dft_matrix_batch_IM;
  hipMalloc((void**)(&dptr_dft_matrix_batch_RE),
             2 * sizeof(__half) * 16 * 16 * 16);
  dptr_dft_matrix_batch_IM =
      dptr_dft_matrix_batch_RE + (16 * 16 * 16);

  ComputeDFTMatrix<<<16, 16*16>>>(dptr_dft_matrix_batch_RE,
                                  dptr_dft_matrix_batch_IM);

  DFTKernel<<<1,32>>>(dptr_results_kernel_RE, dptr_results_kernel_IM,
                      dptr_input_RE, dptr_input_IM, dptr_dft_matrix_batch_RE,
                      dptr_dft_matrix_batch_IM);

  hipMemcpy(data_1.get(), dptr_input_RE, 2*fft_length*sizeof(__half),
             hipMemcpyDeviceToHost);

  WriteResultsToFile("dft_test_kernel.dat", fft_length, data_1.get());

  /*
  for(int i=0; i<fft_length; i++){
    float cpu_re = data_2[i];
    float gpu_re = data_1[i];
    float cpu_im = data_2[i + fft_length];
    float gpu_im = data_1[i + fft_length];
    if ((cpu_re != gpu_re) || (cpu_im != gpu_im)){
      std::cout << "Results of dfts are different! "
                << "CuFFT: " << cpu_re << " " << cpu_im << " Kernel: " << gpu_re
                << " " << gpu_im << std::endl;
      if ((fabs(cpu_re - gpu_re) > 0.01) || (fabs(cpu_im - gpu_im) > 0.01)){
        return false;
      }
    }
  }
  */
  hipfftDestroy(plan);
  hipFree(dptr_cuFFT_out);
  hipFree(dptr_cuFFT_in);
  hipFree(dptr_input_RE);

  return true;
}
