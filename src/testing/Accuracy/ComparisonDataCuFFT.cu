#include "hip/hip_runtime.h"
//Used for the creation of comparission data via CuFFT with varrying precissions

#include <vector>
#include <memory>
#include <iostream>

#include "../DataMaker.cu"

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_fp16.h>
#include <hip/hip_complex.h>

std::unique_ptr<hipfftDoubleComplex> GetComparisionFP64Data(
    float2* dptr_weights, int amount_of_frequencies, int fft_length,
    double normalization_factor){
  //Allocate device memory
  hipfftDoubleComplex* dptr_data;
  hipfftDoubleComplex* dptr_results;
  hipMalloc(&dptr_data, 2 * sizeof(hipfftDoubleComplex) * fft_length);
  dptr_results = dptr_data + fft_length;

  //Produce input data based on weights
  SineSupperposition<hipfftDoubleComplex><<<fft_length / 1024, 1024>>>(
      fft_length, dptr_data, dptr_weights, amount_of_frequencies,
      normalization_factor);

  hipfftHandle plan;
  hipfftResult r;

  r = hipfftCreate(&plan);
  if (r != HIPFFT_SUCCESS) {
    std::cout << "Error! Plan creation failed.\n";
  }

  r = hipfftPlanMany(&plan, 1, &fft_length, nullptr, 1, 1, nullptr, 1, 1,
                    HIPFFT_Z2Z, 1);
  if (r != HIPFFT_SUCCESS) {
    std::cout << "Error! Plan creation failed.";
  }

  r = hipfftExecZ2Z(plan, dptr_data, dptr_results, HIPFFT_FORWARD);
  if (r != HIPFFT_SUCCESS) {
    std::cout << "Error! Plan execution failed.";
  }

  std::unique_ptr<hipfftDoubleComplex> data =
      std::make_unique<hipfftDoubleComplex>(fft_length);

  hipMemcpy(data.get(), dptr_results, fft_length * sizeof(hipfftDoubleComplex),
             hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  hipfftDestroy(plan);
  hipFree(dptr_data);

  return std::move(data);
}

std::unique_ptr<hipfftComplex> GetComparisionFP32Data(
    float2* dptr_weights, int amount_of_frequencies, long long fft_length,
    double normalization_factor){
  //Allocate device memory
  hipfftComplex* dptr_data;
  hipfftComplex* dptr_results;
  hipMalloc(&dptr_data, 2 * sizeof(hipfftComplex) * fft_length);
  dptr_results = dptr_data + fft_length;

  //Produce input data based on weights
  SineSupperposition<hipfftComplex><<<fft_length / 1024, 1024>>>(
      fft_length, dptr_data, dptr_weights, amount_of_frequencies,
      normalization_factor);

  hipfftHandle plan;
  hipfftResult r;

  r = hipfftCreate(&plan);
  if (r != HIPFFT_SUCCESS) {
    std::cout << "Error! Plan creation failed.\n";
  }

  r = hipfftPlanMany(&plan, 1, &fft_length, nullptr, 1, 1, nullptr, 1, 1,
                    HIPFFT_C2C, 1);
  if (r != HIPFFT_SUCCESS) {
    std::cout <<  "Error! Plan creation failed.";
  }

  r = hipfftExecC2C(plan, dptr_data, dptr_results, HIPFFT_FORWARD);
  if (r != HIPFFT_SUCCESS) {
    std::cout <<  "Error! Plan execution failed.";
  }

  std::unique_ptr<hipfftComplex> data =
      std::make_unique<hipfftComplex>(fft_length);

  hipMemcpy(data.get(), dptr_results, fft_length * sizeof(hipfftComplex),
             hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  hipfftDestroy(plan);
  hipFree(dptr_data);

  return std::move(data);
}

std::unique_ptr<__half2> GetComparisionFP16Data(
    float2* dptr_weights, int amount_of_frequencies, long long fft_length,
    double normalization_factor){
  //Allocate device memory
  __half2* dptr_data;
  __half2* dptr_results;
  hipMalloc(&dptr_data, 2 * sizeof(__half2) * fft_length);
  dptr_results = dptr_data + fft_length;

  //Produce input data based on weights
  SineSupperposition<__half2><<<fft_length / 1024, 1024>>>(
      fft_length, dptr_data, dptr_weights, amount_of_frequencies,
      normalization_factor);

  hipfftHandle plan;
  hipfftResult r;

  r = hipfftCreate(&plan);
  if (r != HIPFFT_SUCCESS) {
    std::cout << "Error! Plan creation failed.\n";
  }

  size_t size = 0;
  r = hipfftXtMakePlanMany(plan, 1, &fft_length, nullptr, 1, 1, HIP_C_16F,
                          nullptr, 1, 1, HIP_C_16F, 1, &size, HIP_C_16F);
  if (r != HIPFFT_SUCCESS) {
    std::cout <<  "Error! Plan creation failed.";
  }

  r = hipfftXtExec(plan, dptr_data, dptr_results, HIPFFT_FORWARD);
  if (r != HIPFFT_SUCCESS) {
    std::cout <<  "Error! Plan execution failed.";
  }

  std::unique_ptr<__half2> data = std::make_unique<__half2>(fft_length);

  hipMemcpy(data.get(), dptr_results, fft_length * sizeof(__half2),
             hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  hipfftDestroy(plan);
  hipFree(dptr_data);

  return std::move(data);
}

std::unique_ptr<__half2> GetOurFP16Data(
    float2* dptr_weights, int amount_of_frequencies, long long fft_length,
    double normalization_factor){
  Plan my_plan;
  std::optional<std::string> error_mess = ConfigurePlan(my_plan, fft_length);
  if (error_mess) {
    std::cout << error_mess.value() << std::endl;
  }

  //Check if parameters of plan work given limitations on used device.
  int device_id;
  hipGetDevice(&device_id);
  if (!PlanWorksOnDevice(my_plan, device_id)) {
    std::cout << "Error! Plan imcompatible with used device." << std::endl;
    return false;
  }

  //Allocate device memory
  __half2* dptr_input_data = nullptr;
  __half2* dptr_output_data = nullptr;
  hipMalloc(&dptr_input_data, 2 * sizeof(__half2) * fft_length);
  dptr_output_data = dptr_input_data + fft_length;

  //Produce input data based on weights
  SineSupperposition<__half2><<<fft_length / 1024, 1024>>>(
      fft_length, dptr_input_data, dptr_weights, amount_of_frequencies,
      normalization_factor);

  //Compute the FFT on the device
  error_mess = ComputeFFT(my_plan, dptr_input_data, dptr_output_data,
                          GetMaxNoOptInSharedMem(device_id));
  if (error_mess) {
    std::cout << error_mess.value() << std::endl;
    return false;
  }

  //Needed if data set smaller than 64KB and can be removed otherwise.
  hipDeviceSynchronize();

  //Allocate mem on host for results
  std::unique_ptr<__half2> results = std::make_unique<__half2>(fft_length);

  //Copy results back
  if (hipMemcpy(results.get(),
                 my_plan.results_in_results_ ?
                     dptr_output_data : dptr_input_data,
                 fft_length_ * sizeof(__half2),
                 hipMemcpyDeviceToHost)
       != hipSuccess) {
     std::cout << hipGetErrorString(hipPeekAtLastError()) << std::endl;
     return false;
  }

  //Make sure the results have finished cpying
  hipDeviceSynchronize();

  return std::move(data);
}
