#include "hip/hip_runtime.h"
//Used for the creation of comparission data via CuFFT with varrying precissions
#pragma once

#include <vector>
#include <memory>
#include <iostream>
#include <optional>

#include "../../base/Plan.h"
#include "../../ComputeFFT.h"
#include "../DataMaker.cu"

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_fp16.h>
#include <hip/hip_complex.h>

std::unique_ptr<hipfftDoubleComplex[]> GetComparisionFP64Data(
    float2* dptr_weights, int amount_of_frequencies, int fft_length,
    double normalization_factor){
  //Allocate device memory
  hipfftDoubleComplex* dptr_data;
  hipfftDoubleComplex* dptr_results;
  hipMalloc(&dptr_data, 2 * sizeof(hipfftDoubleComplex) * fft_length);
  dptr_results = dptr_data + fft_length;

  //Produce input data based on weights
  SineSupperposition<hipfftDoubleComplex><<<fft_length / 1024, 1024>>>(
      fft_length, dptr_data, dptr_weights, amount_of_frequencies,
      normalization_factor);

  hipfftHandle plan;
  hipfftResult r;

  r = hipfftCreate(&plan);
  if (r != HIPFFT_SUCCESS) {
    std::cout << "Error! Plan creation failed.\n" << std::endl;
  }

  r = hipfftPlanMany(&plan, 1, &fft_length, nullptr, 1, 1, nullptr, 1, 1,
                    HIPFFT_Z2Z, 1);
  if (r != HIPFFT_SUCCESS) {
    std::cout << "Error! Plan creation failed." << std::endl;
  }

  r = hipfftExecZ2Z(plan, dptr_data, dptr_results, HIPFFT_FORWARD);
  if (r != HIPFFT_SUCCESS) {
    std::cout << "Error! Plan execution failed." << std::endl;
  }

  std::unique_ptr<hipfftDoubleComplex[]> data =
      std::make_unique<hipfftDoubleComplex[]>(fft_length);

  hipMemcpy(data.get(), dptr_results, fft_length * sizeof(hipfftDoubleComplex),
             hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  hipfftDestroy(plan);
  hipFree(dptr_data);

  return data;
}

std::unique_ptr<hipfftComplex[]> GetComparisionFP32Data(
    float2* dptr_weights, int amount_of_frequencies, int fft_length,
    double normalization_factor){
  //Allocate device memory
  hipfftComplex* dptr_data;
  hipfftComplex* dptr_results;
  hipMalloc(&dptr_data, 2 * sizeof(hipfftComplex) * fft_length);
  dptr_results = dptr_data + fft_length;

  //Produce input data based on weights
  SineSupperposition<hipfftComplex><<<fft_length / 1024, 1024>>>(
      fft_length, dptr_data, dptr_weights, amount_of_frequencies,
      normalization_factor);

  hipfftHandle plan;
  hipfftResult r;

  r = hipfftCreate(&plan);
  if (r != HIPFFT_SUCCESS) {
    std::cout << "Error! Plan creation failed.\n" << std::endl;
  }

  r = hipfftPlanMany(&plan, 1, &fft_length, nullptr, 1, 1, nullptr, 1, 1,
                    HIPFFT_C2C, 1);
  if (r != HIPFFT_SUCCESS) {
    std::cout <<  "Error! Plan creation failed." << std::endl;
  }

  r = hipfftExecC2C(plan, dptr_data, dptr_results, HIPFFT_FORWARD);
  if (r != HIPFFT_SUCCESS) {
    std::cout <<  "Error! Plan execution failed." << std::endl;
  }

  std::unique_ptr<hipfftComplex[]> data =
      std::make_unique<hipfftComplex[]>(fft_length);

  hipMemcpy(data.get(), dptr_results, fft_length * sizeof(hipfftComplex),
             hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  hipfftDestroy(plan);
  hipFree(dptr_data);

  return data;
}

std::unique_ptr<__half2[]> GetComparisionFP16Data(
    float2* dptr_weights, int amount_of_frequencies, long long fft_length,
    double normalization_factor){
  //Allocate device memory
  __half2* dptr_data;
  __half2* dptr_results;
  hipMalloc(&dptr_data, 2 * sizeof(__half2) * fft_length);
  dptr_results = dptr_data + fft_length;

  //Produce input data based on weights
  SineSupperposition<__half2><<<fft_length / 1024, 1024>>>(
      fft_length, dptr_data, dptr_weights, amount_of_frequencies,
      normalization_factor);

  hipfftHandle plan;
  hipfftResult r;

  r = hipfftCreate(&plan);
  if (r != HIPFFT_SUCCESS) {
    std::cout << "Error! Plan creation failed.\n" << std::endl;
  }

  size_t size = 0;
  r = hipfftXtMakePlanMany(plan, 1, &fft_length, nullptr, 1, 1, HIP_C_16F,
                          nullptr, 1, 1, HIP_C_16F, 1, &size, HIP_C_16F);
  if (r != HIPFFT_SUCCESS) {
    std::cout <<  "Error! Plan creation failed." << std::endl;
  }

  r = hipfftXtExec(plan, dptr_data, dptr_results, HIPFFT_FORWARD);
  if (r != HIPFFT_SUCCESS) {
    std::cout <<  "Error! Plan execution failed." << std::endl;
  }

  std::unique_ptr<__half2[]> data = std::make_unique<__half2[]>(fft_length);

  hipMemcpy(data.get(), dptr_results, fft_length * sizeof(__half2),
             hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  hipfftDestroy(plan);
  hipFree(dptr_data);

  return data;
}

template<typename Integer>
std::unique_ptr<__half2[]> GetOurFP16Data(
    float2* dptr_weights, int amount_of_frequencies, long long fft_length,
    double normalization_factor){
  std::optional<Plan> possible_plan = MakePlan(fft_length);
  Plan my_plan;
  if (possible_plan) {
    my_plan = possible_plan.value();
  } else {
    std::cout << "Plan creation failed" << std::endl;
  }

  //Check if parameters of plan work given limitations on used device.
  int device_id;
  hipGetDevice(&device_id);
  if (!PlanWorksOnDevice(my_plan, device_id)) {
    std::cout << "Error! Plan imcompatible with used device." << std::endl;
  }

  //Allocate device memory
  __half2* dptr_input_data = nullptr;
  __half2* dptr_output_data = nullptr;
  hipMalloc(&dptr_input_data, 2 * sizeof(__half2) * fft_length);
  dptr_output_data = dptr_input_data + fft_length;

  //Produce input data based on weights
  SineSupperposition<__half2><<<fft_length / 1024, 1024>>>(
      fft_length, dptr_input_data, dptr_weights, amount_of_frequencies,
      normalization_factor);

  //Compute the FFT on the device
  std::optional<std::string> error_mess =
      ComputeFFT<Integer>(my_plan, dptr_input_data, dptr_output_data,
                          GetMaxNoOptInSharedMem(device_id));
  if (error_mess) {
    std::cout << error_mess.value() << std::endl;
  }

  //Needed if data set smaller than 64KB and can be removed otherwise.
  hipDeviceSynchronize();

  //Allocate mem on host for results
  std::unique_ptr<__half2[]> results = std::make_unique<__half2[]>(fft_length);

  //Copy results back
  if (hipMemcpy(results.get(),
                 my_plan.results_in_results_ ?
                     dptr_output_data : dptr_input_data,
                 fft_length * sizeof(__half2),
                 hipMemcpyDeviceToHost)
       != hipSuccess) {
     std::cout << hipGetErrorString(hipPeekAtLastError()) << std::endl;
  }

  //Make sure the results have finished cpying
  hipDeviceSynchronize();

  return results;
}
