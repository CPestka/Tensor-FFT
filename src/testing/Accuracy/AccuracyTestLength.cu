#include "hip/hip_runtime.h"
//Accuracy benchmark for a varrying length but with fixed frequency bandwidth.
#include <vector>
#include <memory>

#include "../DataMaker.cu"
#include "../WeightMaker.h"
#include "ComputeError.h"
#include "../FileWriter.h"

template <typename Integer>
Integer ExactPowerOf2(const int exponent){
  if (exponent < 0) {
    std::cout << "Error! Negative exponent not allowed." << std::endl;
  }

  Integer result = 1;
  for(int i=0; i<exponent; i++){
    result *=2;
  }
  return result;
}

template <typename Integer>
double GetNormalizationFactor(double normalization_target, float2* dptr_weights,
                              int amount_of_frequencies, Integer fft_length){
  hipfftDoubleComplex* dptr_data;
  hipMalloc(&dptr_data, sizeof(hipfftDoubleComplex) * fft_length);
  //Produce input data based on weights
  SineSupperposition<int,hipfftDoubleComplex><<<fft_length / 1024, 1024>>>(
      fft_length, dptr_data, dptr_weights, amount_of_frequencies, 1.0);

  std::unique_ptr<hipfftDoubleComplex[]> data =
      std::make_unique<hipfftDoubleComplex[]>(fft_length);

  hipMemcpy(data.get(), dptr_data, fft_length * sizeof(hipfftDoubleComplex),
             hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  hipFree(dptr_data);

  return (MaxValue<Integer,hipfftDoubleComplex>(data.get(), fft_length) /
          normalization_target);
}

int main(){
  int fft_length_min_log2 = 8;
  int fft_length_max_log2 = 28;
  int amount_of_frequencies = 16*16*16*16*16;
  std::vector<double> normalize_to;

  std::unique_ptr<float2[]> weights =
      std::make_unique<float2[]>(amount_of_frequencies);
  SetRandomWeights(weights.get(), amount_of_frequencies, 42*42);
  float2* dptr_weights = nullptr;
  hipMalloc(&dptr_weights, sizeof(float2) * amount_of_frequencies);
  hipMemcpy(dptr_weights, weights.get(),
             sizeof(float2) * amount_of_frequencies, hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  std::vector<int> fft_lengths;
  std::vector<Errors> errors;
  std::vector<int> amount_of_frequencies_vec;

  for(int i=fft_length_min_log2; i<=fft_length_max_log2; i++){
    normalize_to.push_back(1.0);
    fft_lengths.push_back(ExactPowerOf2<int>(i));
    std::cout << fft_lengths.back() << std::endl;
    double normalization_factor =
        GetNormalizationFactor<int>(normalize_to.back(), dptr_weights,
                                    amount_of_frequencies, fft_lengths.back());
    errors.push_back(ComputeOurVsFp64Errors<int>(static_cast<int>(fft_lengths.back()),
        dptr_weights, amount_of_frequencies, normalization_factor));
    amount_of_frequencies_vec.push_back(amount_of_frequencies);
  }

  WriteAccuracyToFile("AccTest_our_N.dat", normalize_to, fft_lengths, errors,
                      amount_of_frequencies_vec);

  hipFree(dptr_weights);
}
