#include "hip/hip_runtime.h"
//Accuracy benchmark for a varrying length but with fixed frequency bandwidth.
#include <vector>
#include <memory>

#include "../DataMaker.cu"
#include "../WeightMaker.h"
#include "ComputeError.h"
#include "../FileWriter.h"

template <typename Integer>
Integer ExactPowerOf2(const int exponent){
  if (exponent < 0) {
    std::cout << "Error! Negative exponent not allowed." << std::endl;
  }

  Integer result = 1;
  for(int i=0; i<exponent; i++){
    result *=2;
  }
  return result;
}

template <typename Integer>
double GetNormalizationFactor(double normalization_target, float2* dptr_weights,
                              int amount_of_frequencies, Integer fft_length){
  hipfftDoubleComplex* dptr_data;
  hipMalloc(&dptr_data, sizeof(hipfftDoubleComplex) * fft_length);
  //Produce input data based on weights
  SineSupperposition<int,hipfftDoubleComplex><<<fft_length / 1024, 1024>>>(
      fft_length, dptr_data, dptr_weights, amount_of_frequencies, 1.0);

  std::unique_ptr<hipfftDoubleComplex[]> data =
      std::make_unique<hipfftDoubleComplex[]>(fft_length);

  hipMemcpy(data.get(), dptr_data, fft_length * sizeof(hipfftDoubleComplex),
             hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  hipFree(dptr_data);

  return (normalization_target /
          MaxValue<Integer,hipfftDoubleComplex>(data.get(), fft_length));
}

// int main(){
//   int fft_length_min_log16 = 3;
//   int fft_length_max_log16 = 7;
//   int amount_of_frequencies = 256;
//   double normalize_to = 1.0;
//
//   std::unique_ptr<float2[]> weights =
//       std::make_unique<float2[]>(amount_of_frequencies);
//   SetRandomWeights(weights.get(), amount_of_frequencies, 42*42);
//   float2* dptr_weights = nullptr;
//   hipMalloc(&dptr_weights, sizeof(float2) * amount_of_frequencies);
//   hipMemcpy(dptr_weights, weights.get(),
//              sizeof(float2) * amount_of_frequencies, hipMemcpyHostToDevice);
//   hipDeviceSynchronize();
//
//   std::vector<int> fft_lengths;
//   std::vector<Errors> errors;
//   std::vector<int> amount_of_frequencies_vec;
//
//   for(int i=fft_length_min_log16; i<=fft_length_max_log16; i++){
//     fft_lengths.push_back(ExactPowerOf2<int>(i*4));
//     std::cout << fft_lengths.back() << std::endl;
//     double normalization_factor =
//         GetNormalizationFactor<int>(normalize_to, weights.get(),
//                                     amount_of_frequencies, fft_lengths.back());
//     errors.push_back(ComputeOurVsFp64Errors<int>(fft_lengths.back(),
//         dptr_weights, amount_of_frequencies, normalization_factor));
//     amount_of_frequencies_vec.push_back(amount_of_frequencies);
//   }
//
//   WriteAccuracyToFile("AccuracyTest.dat", normalize_to, fft_lengths, errors,
//                       amount_of_frequencies_vec);
//
//   hipFree(dptr_weights);
// }

int main(){
    int amount_of_frequencies = 256;
    double normalize_to = 1.0;

    std::unique_ptr<float2[]> weights =
        std::make_unique<float2[]>(amount_of_frequencies);
    SetRandomWeights(weights.get(), amount_of_frequencies, 42*42);

    float2* dptr_weights;
    hipMalloc(&dptr_weights, sizeof(float2) * amount_of_frequencies);
    hipMemcpy(dptr_weights, weights.get(),
               sizeof(float2) * amount_of_frequencies, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    std::vector<int> fft_lengths;
    fft_lengths.push_back(16*16*16);
    fft_lengths.push_back(16*16*16*16);
    fft_lengths.push_back(16*16*16*16*16);
    fft_lengths.push_back(16*16*16*16*16 * 16);
    fft_lengths.push_back(16*16*16*16*16 * 16*16);

    std::vector<Errors> errors;
    std::vector<int> amount_of_frequencies_vec;
    amount_of_frequencies_vec.push_back(amount_of_frequencies);

    for(int i=0; i<5; i++){
      Errors test_err = ComputeOurVsFp64Errors<int>(fft_lengths[i], dptr_weights, amount_of_frequencies, 1.0);
      std::cout << test_err.MaxDiv << std::endl;
    }

    return true;
}
