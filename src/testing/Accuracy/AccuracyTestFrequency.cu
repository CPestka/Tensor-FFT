#include "hip/hip_runtime.h"
//Accuracy benchmark for a varrying frequency bandwidth but with fixed length.
#include <vector>
#include <memory>

#include "../DataMaker.cu"
#include "../WeightMaker.h"
#include "ComputeError.h"
#include "../FileWriter.h"

template <typename Integer>
Integer ExactPowerOf2(const int exponent){
  if (exponent < 0) {
    std::cout << "Error! Negative exponent not allowed." << std::endl;
  }

  Integer result = 1;
  for(int i=0; i<exponent; i++){
    result *=2;
  }
  return result;
}

template <typename Integer>
double GetNormalizationFactor(double normalization_target, float2* dptr_weights,
                              int amount_of_frequencies, Integer fft_length){
  hipfftDoubleComplex* dptr_data;
  hipMalloc(&dptr_data, sizeof(hipfftDoubleComplex) * fft_length);
  //Produce input data based on weights
  SineSupperposition<int,hipfftDoubleComplex><<<fft_length / 1024, 1024>>>(
      fft_length, dptr_data, dptr_weights, amount_of_frequencies, 1.0);

  std::unique_ptr<hipfftDoubleComplex[]> data =
      std::make_unique<hipfftDoubleComplex[]>(fft_length);

  hipMemcpy(data.get(), dptr_data, fft_length * sizeof(hipfftDoubleComplex),
             hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  return (normalization_target /
          MaxValue<Integer,hipfftDoubleComplex>(data.get(), fft_length));
}

int main(){
  int fft_length = 16*16*16*16*16 * 16*16;
  int max_frequencies = fft_length;
  int frequency_steps = 32;
  int frequency_increment = fft_length / frequency_steps;

  double normalize_to = 1.0;

  std::vector<int> fft_lengths;
  std::vector<Errors> errors;
  std::vector<int> amount_of_frequencies_vec;

  std::unique_ptr<float2[]> weights =
      std::make_unique<float2[]>(max_frequencies);
  float2* dptr_weights = nullptr;
  hipMalloc(&dptr_weights, sizeof(float2) * max_frequencies);

  for(int i=1; i<=frequency_steps; i++){
    SetRandomWeights(weights.get(), frequency_increment * i, 42*42);
    hipMemcpy(dptr_weights, weights.get(),
               sizeof(float2) * frequency_increment * i,
               hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    double normalization_factor =
        GetNormalizationFactor<int>(normalize_to, dptr_weights,
                                    frequency_increment * i, fft_length);

    fft_lengths.push_back(fft_length);
    amount_of_frequencies_vec.push_back(frequency_increment * i);

    errors.push_back(ComputeOurVsFp64Errors<int>(fft_lengths.back(),
        dptr_weights, frequency_increment * i, normalization_factor));

    std::cout << frequency_increment * i << std::endl;

  }

  WriteAccuracyToFile("AccTest_our_nu.dat", normalize_to, fft_lengths, errors,
                      amount_of_frequencies_vec);

  hipFree(dptr_weights);
}
