#include "hip/hip_runtime.h"
//Used to profile the function ComputeFFT
#include <iostream>
#include <vector>
#include <optional>
#include <string>
#include <memory>

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "TestingDataCreation.h"
#include "FileWriter.h"

int main(){
  long long fft_length = 16*16*16;

  std::vector<float> weights_RE { 1.0, 0.7, 0.5, 0.2, 1.0, 0.9, 0.4 };
  std::vector<float> weights_IM { 1.0, 0.3, 0.2, 0.4, 0.1, 0.5, 0.9 };
  std::unique_ptr<__half2[]> data =
      CreateSineSuperpostionH2(fft_length, weights_RE, weights_IM);

  __half2* dptr_data;
  __half2* dptr_results;
  hipMalloc(&dptr_data, sizeof(__half2) * fft_length);
  hipMalloc(&dptr_results, sizeof(__half2) * fft_length);

  hipfftHandle plan;
  hipfftResult r;

  r = hipfftCreate(&plan);
  if (r != HIPFFT_SUCCESS) {
    std::cout << "Error! Plan creation failed." << std::endl;
    return false;
  }

  size_t size = 0;
  r = hipfftXtMakePlanMany(plan, 1, &fft_length, nullptr, 1, 1,
                          HIP_C_16F, nullptr, 1, 1, HIP_C_16F, 1, &size,
                          HIP_C_16F);
  //r = hipfftPlan1d(&plan, fft_length, HIPFFT_C2C, 0);

  if (r != HIPFFT_SUCCESS) {
    std::cout << "Error! Plan creation failed." << std::endl;
    return false;
  }

  hipMemcpy(dptr_data, data.get(), fft_length * sizeof(__half2),
             hipMemcpyHostToDevice);

  r = hipfftXtExec(plan, dptr_data, dptr_results, HIPFFT_FORWARD);
  //r = hipfftExecC2C(plan, dptr_data, dptr_results, HIPFFT_FORWARD);
  if (r != HIPFFT_SUCCESS) {
    std::cout << "Error! Plan execution failed." << std::endl;
    return false;
  }

  hipDeviceSynchronize();

  //Write results to file
  WriteResultsToFileHalf2("example_results.dat", fft_length, data.get());

  hipfftDestroy(plan);
  hipFree(dptr_results);
  hipFree(dptr_data);

  return true;
}
