#include "hip/hip_runtime.h"
//Used to profile the function ComputeFFT
#include <iostream>
#include <vector>
#include <optional>
#include <string>
#include <memory>

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "TestingDataCreation.h"
#include "FileWriter.h"

int main(){
  long long fft_length = 16*16*16*16*16 * 16*2;

  std::vector<float> weights_RE { 1.0, 0.7, 0.5, 0.2, 0.3, 0.7, 0.8 };
  std::vector<float> weights_IM { 1.0, 0.3, 0.2, 0.4, 0.9, 0.1, 0.6 };
  std::unique_ptr<__half2[]> data =
      CreateSineSuperpostionH2GPU(fft_length, weights_RE, weights_IM, 7);

  __half2* dptr_data;
  __half2* dptr_results;
  hipMalloc(&dptr_data, sizeof(__half2) * fft_length);
  hipMalloc(&dptr_results, sizeof(__half2) * fft_length);

  hipfftHandle plan;
  hipfftResult r;

  r = hipfftCreate(&plan);
  if (r != HIPFFT_SUCCESS) {
    std::cout << "Error! Plan creation failed." << std::endl;
    return false;
  }

  size_t size = 0;
  r = hipfftXtMakePlanMany(plan, 1, &fft_length, nullptr, 1, 1,
                          HIP_C_16F, nullptr, 1, 1, HIP_C_16F, 1, &size,
                          HIP_C_16F);
  //r = hipfftPlan1d(&plan, fft_length, HIPFFT_Z2Z, 1);

  if (r != HIPFFT_SUCCESS) {
    std::cout << "Error! Plan creation failed." << std::endl;
    return false;
  }

  hipMemcpy(dptr_data, data.get(), fft_length * sizeof(__half2),
             hipMemcpyHostToDevice);

  r = hipfftXtExec(plan, dptr_data, dptr_results, HIPFFT_FORWARD);
  //r = hipfftExecZ2Z(plan, dptr_data, dptr_results, HIPFFT_FORWARD);
  if (r != HIPFFT_SUCCESS) {
    std::cout << "Error! Plan execution failed." << std::endl;
    return false;
  }

  hipMemcpy(data.get(), dptr_results, fft_length * sizeof(__half2),
             hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  //Write results to file
  //WriteResultsToFileDouble2("example_results.dat", fft_length, data.get());

  hipfftDestroy(plan);
  hipFree(dptr_results);
  hipFree(dptr_data);

  return true;
}
