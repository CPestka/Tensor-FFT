#include "hip/hip_runtime.h"
//Used to benchmark the function ComputeFFT
#include <iostream>
#include <vector>
#include <optional>
#include <string>
#include <memory>

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "TestingDataCreation.cu"

int main(){
  long long fft_length = 16*16*16*16*16*16*2*2*2;

  std::vector<float> weights;
  weights.push_back(1.0);
  std::unique_ptr<__half2[]> data =
      CreateSineSuperpostionH2(fft_length,  weights);

  __half2* dptr_data;
  __half2* dptr_results;
  hipMalloc(&dptr_data, sizeof(__half2) * fft_length);
  hipMalloc(&dptr_results, sizeof(__half2) * fft_length);

  hipfftHandle plan;
  hipfftResult r;

  r = hipfftCreate(&plan);
  if (r != HIPFFT_SUCCESS) {
    std::cout << "Error! Plan creation failed." << std::endl;
    return false;
  }

  size_t size = 0;
  r = hipfftXtMakePlanMany(plan, 1, &fft_length, nullptr, 1, 1,
                          HIP_C_16F, nullptr, 1, 1, HIP_C_16F, 1, &size,
                          HIP_C_16F);
  if (r != HIPFFT_SUCCESS) {
    std::cout << "Error! Plan creation failed." << std::endl;
    return false;
  }

  hipMemcpy(dptr_data, data.get(), fft_length * sizeof(__half2),
             hipMemcpyHostToDevice);

  r = hipfftXtExec(plan, dptr_data, dptr_results, HIPFFT_FORWARD);
  if (r != HIPFFT_SUCCESS) {
    std::cout << "Error! Plan execution failed." << std::endl;
    return false;
  }

  hipDeviceSynchronize();

  hipfftDestroy(plan);
  hipFree(dptr_results);
  hipFree(dptr_data);

  return true;
}
