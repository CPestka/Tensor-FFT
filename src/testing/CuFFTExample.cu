//Used to profile the function ComputeFFT
#include <iostream>
#include <vector>
#include <optional>
#include <string>
#include <memory>

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "TestingDataCreation.h"
#include "FileWriter.h"

int main(){
  long long fft_length = 16*16*16;

  std::vector<float> weights_RE { 1.0, 0.7, 0.5, 0.2, 0.3, 0.7, 0.8 };
  std::vector<float> weights_IM { 1.0, 0.3, 0.2, 0.4, 0.9, 0.1, 0.6 };
  std::unique_ptr<hipfftDoubleComplex[]> data =
      CreateSineSuperpostionDouble(fft_length, weights_RE, weights_IM);

  hipfftDoubleComplex* dptr_data;
  hipfftDoubleComplex* dptr_results;
  hipMalloc(&dptr_data, sizeof(hipfftDoubleComplex) * fft_length);
  hipMalloc(&dptr_results, sizeof(hipfftDoubleComplex) * fft_length);

  hipfftHandle plan;
  hipfftResult r;

  r = hipfftCreate(&plan);
  if (r != HIPFFT_SUCCESS) {
    std::cout << "Error! Plan creation failed." << std::endl;
    return false;
  }

  size_t size = 0;
  // r = hipfftXtMakePlanMany(plan, 1, &fft_length, nullptr, 1, 1,
  //                         HIP_C_16F, nullptr, 1, 1, HIP_C_16F, 1, &size,
  //                         HIP_C_16F);
  r = hipfftPlan1d(&plan, fft_length, HIPFFT_Z2Z, 1);

  if (r != HIPFFT_SUCCESS) {
    std::cout << "Error! Plan creation failed." << std::endl;
    return false;
  }

  hipMemcpy(dptr_data, data.get(), fft_length * sizeof(hipfftDoubleComplex),
             hipMemcpyHostToDevice);

  // r = hipfftXtExec(plan, dptr_data, dptr_results, HIPFFT_FORWARD);
  r = hipfftExecC2C(plan, dptr_data, dptr_results, HIPFFT_FORWARD);
  if (r != HIPFFT_SUCCESS) {
    std::cout << "Error! Plan execution failed." << std::endl;
    return false;
  }

  hipMemcpy(data.get(), dptr_results, fft_length * sizeof(hipfftDoubleComplex),
             hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  //Write results to file
  WriteResultsToFileDouble2("example_results.dat", fft_length, data.get());

  hipfftDestroy(plan);
  hipFree(dptr_results);
  hipFree(dptr_data);

  return true;
}
