#include "hip/hip_runtime.h"

#include <vector>
#include <memory>
#include <iostream>
#include <string>
#include <optional>
#include <cstdint>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "../DataMaker.cu"
#include "../WeightMaker.h"
#include "../FileWriter.h"
#include "../../base/Plan.h"
#include "../../base/ComputeFFT.h"
#include "../Timer.h"

double GetAverage(std::vector<double> data){
  double tmp = 0;
  for(int i=0; i<static_cast<int>(data.size()); i++){
    tmp += data[i];
  }

  return (tmp/static_cast<double>(data.size()));
}

double GetRMS(std::vector<double> data, double average){
  double tmp = 0;

  for(int i=0; i<static_cast<int>(data.size()); i++){
    double tmp2 = data[i] - average;
    tmp += (tmp2 * tmp2);
  }

  return sqrt(tmp / static_cast<double>(data.size()));
}

int main(){
  constexpr int min_fft_length = 16*16*16;
  constexpr int max_fft_length = 16*16*16*16*16 * 16*16;
  int max_frequencies = 10;
  int samples = 1000;
  int warmup_samples = 20;
  int total_samples = samples + warmup_samples;

  std::vector<BatchResult> results;

  int64_t current_fft_length = min_fft_length;

  while(current_fft_length <= max_fft_length){
    std::cout << "Current fft_length: " << current_fft_length << std::endl;

    std::unique_ptr<float2[]> weights =
        std::make_unique<float2[]>(max_frequencies);
    float2* dptr_weights = nullptr;
    hipMalloc(&dptr_weights, sizeof(float2) * max_frequencies);

    SetRandomWeights(weights.get(), max_frequencies, 42*42);
    hipMemcpy(dptr_weights, weights.get(),
               sizeof(float2) * max_frequencies,
               hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    __half2* dptr_data;
    __half2* dptr_results;
    hipMalloc((void**)(&dptr_data), static_cast<int>(sizeof(__half2) * current_fft_length));
    hipMalloc((void**)(&dptr_results), static_cast<int>(sizeof(__half2) * current_fft_length));

    std::optional<Plan> possible_plan = MakePlan(current_fft_length);
    Plan my_plan;
    if (possible_plan) {
      my_plan = possible_plan.value();
    } else {
      std::cout << "Plan creation failed" << std::endl;
      return false;
    }

    std::vector<double> runtimes;

    for(int i=0; i<total_samples; i++){
      double runtime;

      SineSupperposition<int,__half2><<<current_fft_length / 1024, 1024>>>(
          static_cast<int>(current_fft_length), dptr_data, dptr_weights, max_frequencies, 1.0);

      hipDeviceSynchronize();

      IntervallTimer timer;

      ComputeFFT<int>(my_plan, dptr_data, dptr_results);

      hipDeviceSynchronize();

      runtime = static_cast<double>(timer.getTimeInNanoseconds());

      if(i >= warmup_samples) {
        runtimes.push_back(runtime);
      }
    }

    BatchResult current_result;
    current_result.Average_ = GetAverage(runtimes);
    current_result.RMS_ = GetRMS(runtimes, current_result.Average_);
    current_result.fft_length_ = current_fft_length;

    results.push_back(current_result);

    hipFree(dptr_results);
    hipFree(dptr_data);

    current_fft_length = current_fft_length * 16;
  }

  std::cout << "Finished Benchmark" << std::endl;

  WriteBenchResultsToFile("BenchOur.dat", results);

  return true;
}
