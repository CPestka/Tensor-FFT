#include "hip/hip_runtime.h"
//Used to benchmark the function ComputeFFT
#include <iostream>
#include <vector>
#include <optional>
#include <string>
#include <memory>

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "../TestingDataCreation.cu"
#include "../FileWriter.cu"
#include "../Timer.h"
#include "../../base/ComputeFFT.cu"
#include "../../base/Plan.cpp"

double ComputeAverage(std::vector<double> data){
  double tmp = 0;
  for(int i=0; i<static_cast<int>(data.size()); i++){
    tmp += data[i];
  }
  return (tmp / (static_cast<double>(data.size()) - 1));
}

double ComputeSigma(std::vector<double> data, double average){
  double tmp = 0;
  for(int i=0; i<static_cast<int>(data.size()); i++){
    double tmp_1 = data[i] - average;
    tmp += (tmp_1 * tmp_1);
  }
  return sqrt(tmp / (static_cast<double>(data.size()) - 1));
}

int main(){
  int log_length_max = 12;
  int sample_size = 20;
  int warmup_samples = 5;

  std::vector<long long> fft_length;
  std::vector<double> avg_runtime;
  std::vector<double> sigma_runtime;

  std::optional<std::string> error_mess;

  int length = 16 * 8;
  for(int i=8; i<=log_length_max; i++){
    length = length * 2;
    fft_length.push_back(length);
    std::cout << "Starting fft length: " << length << std::endl;

    std::vector<float> weights;
    weights.push_back(1.0);
    std::unique_ptr<__half2[]> data =
        CreateSineSuperpostionH2(fft_length.back(),  weights);

    std::vector<double> runtime;

    __half2* dptr_data;
    __half2* dptr_results;
    hipMalloc(&dptr_data, sizeof(__half2) * fft_length.back());
    hipMalloc(&dptr_results, sizeof(__half2) * fft_length.back());

    hipfftHandle plan;
    hipfftResult r;

    r = hipfftCreate(&plan);
    if (r != HIPFFT_SUCCESS) {
      std::cout << "Error! Plan creation failed." << std::endl;
      return false;
    }

    size_t size = 0;
    r = hipfftXtMakePlanMany(plan, 1, &fft_length.back(), nullptr, 1, 1,
                            HIP_C_16F, nullptr, 1, 1, HIP_C_16F, 1, &size,
                            HIP_C_16F);
    if (r != HIPFFT_SUCCESS) {
      std::cout << "Error! Plan creation failed." << std::endl;
      return false;
    }

    for(int k=0; k<sample_size + warmup_samples; k++){
      hipMemcpy(dptr_data, data.get(), fft_length.back() * sizeof(__half2),
                 hipMemcpyHostToDevice);

      hipDeviceSynchronize();

      IntervallTimer computation_time;

      r = hipfftXtExec(plan, dptr_data, dptr_results, HIPFFT_FORWARD);
      if (r != HIPFFT_SUCCESS) {
        std::cout << "Error! Plan execution failed." << std::endl;
        return false;
      }

      hipDeviceSynchronize();

      if (k >= warmup_samples) {
        runtime.push_back(computation_time.getTimeInNanoseconds());
      }
    }

    avg_runtime.push_back(ComputeAverage(runtime));
    sigma_runtime.push_back(ComputeSigma(runtime, avg_runtime.back()));

    hipfftDestroy(plan);
    hipFree(dptr_results);
    hipFree(dptr_data);
  }

  WriteBenchResultsToFile(avg_runtime, sigma_runtime, fft_length,
                          std::to_string(sample_size) + "_cuFFT");
  return true;
}
