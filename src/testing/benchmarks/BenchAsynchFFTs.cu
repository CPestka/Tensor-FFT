#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdint>
#include <string>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "../base/Plan.cpp"
#include "ComputeFFTs.cu"
#include "TestingDataCreation.cu"
#include "../base/Timer.h"

int main(){
  int transpose_blocksize = 256;
  int dft_warps_per_block;
  int r16_warps_per_block;
  int r2_blocksize = 256;

  int ffts_per_call = 40;
  int sample_size = 100;
  int log2_upper_limit = 31; //30 approx 10^9
  std::vector<std::vector<int64_t>> run_times;

  int fft_length = 16*16*16; //i.e. 2^12
  for(int j=12; j<=log2_upper_limit; j++){
    std::vector<int64_t> tmp;
    for(int i=0; i<sample_size; i++){
      std::vector<Plan> my_plan;
      if (j<15) {
        dft_warps_per_block = 1;
        r16_warps_per_block = 1;
      } else {
        dft_warps_per_block = 4;
        r16_warps_per_block = 4;
      }
      for(int k=0; k<ffts_per_call; k++){
        auto tmp = CreatePlan(fft_length, transpose_blocksize, dft_warps_per_block,
                              r16_warps_per_block, r2_blocksize);
        if (tmp.has_value) {
          my_plan.push_back(tmp.value);
        } else {
          return false;
        }
      }

      std::vector<float> weights;
      weights.push_back(2.0);
      weights.push_back(1.4);
      std::vector<__half*> data;
      for(int k=0; k<amount_of_ffts; k++){
        data.push_back(CreateSineSuperpostion(fft_length, weights).get());
      }

      IntervallTimer timer;
      auto error_mess = ComputeFFTs(my_plan, data);
      tmp.push_back(timer.getTimeInMicroseconds() / ffts_per_call);
    }
    run_times.push_back(tmp);
    fft_length = fft_length * 2;
  }

  std::vector<double> average;
  std::vector<double> std_dev;

  for(int i=0; i<run_times.size(); i++){
    double tmp = 0;
    for(int j=0; j<run_times[i].size(); j++){
      tmp += (run_times[i][j] / 1000000);
    }
    tmp = tmp / run_times[i].size();
    average.push_back(tmp);
  }

  for(int i=0; i<run_times.size(); i++){
    double tmp = 0;
    for(int j=0; j<run_times[i].size(); j++){
      double tmp1 = (run_times[i][j] / 1000000) - average[i];
      tmp += (tmp1 * tmp1);
    }
    tmp = tmp / (run_times[i].size() - 1);
    std_dev.push_back(tmp);
  }

  WriteBenchResultsToFile(average, std_dev,
      std::to_string(sample_size) + "Asynch" + std::to_string(ffts_per_call));
}
