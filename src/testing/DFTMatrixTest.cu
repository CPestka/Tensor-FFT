#include "hip/hip_runtime.h"
//Used to test correctness of dft matrix computed on the gpu

#pragma once

#include <iostream>
#include <cstdint>
#include <string>
#include <vector>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "../base/ComputeDFTMatrix.cu"

bool dft_matrix16_test(){
  float dft_matrix_cpu_RE[16][16];
  float dft_matrix_cpu_IM[16][16];

  std::cout << "PI = " << M_PI << std::endl;

  for(int i=0; i<16; i++){
    for(int j=0; j<16; j++){
      dft_matrix_cpu_RE[j][i] = cos((2*M_PI*i*j)/16);
      dft_matrix_cpu_IM[j][i] = -sin((2*M_PI*i*j)/16);
    }
  }

  std::unique_ptr<__half[]> dft_matrix_gpu_RE =
      std::make_unique<__half[]>(16*16);
  std::unique_ptr<__half[]> dft_matrix_gpu_IM =
      std::make_unique<__half[]>(16*16);

  __half* dptr_dft_matrix_gpu_RE;
  __half* dptr_dft_matrix_gpu_IM;
  hipMalloc((void**)(&dptr_dft_matrix_gpu_RE), sizeof(__half)*16*16);
  hipMalloc((void**)(&dptr_dft_matrix_gpu_IM), sizeof(__half)*16*16);

  ComputeDFTMatrix<<<1,16>>>(dptr_dft_matrix_gpu_RE, dptr_dft_matrix_gpu_IM);

  hipMemcpy(dft_matrix_gpu_RE.get(), dptr_dft_matrix_gpu_RE,
             16*16*sizeof(__half), hipMemcpyDeviceToHost);
  hipMemcpy(dft_matrix_gpu_IM.get(), dptr_dft_matrix_gpu_IM,
             16*16*sizeof(__half), hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  for(int j=0; j<16; j++){
    for(int i=0; i<16; i++){
      double gpu_RE = dft_matrix_gpu_RE[i + 16*j];
      double gpu_IM = dft_matrix_gpu_IM[i + 16*j];

      if ((fabs((dft_matrix_cpu_RE[j][i] - gpu_RE)) > 0.0001) ||
          (fabs((dft_matrix_cpu_IM[j][i] - gpu_IM)) > 0.0001)){
        std::cout << "DFT matrix on CPU and GPU are different!"
                  << std::endl;
        return false;
      }
    }
  }

  std::cout << "CPU_RE:" << std::endl;
  for(int j=0; j<16; j++){
    for(int i=0; i<16; i++){
      std::cout << dft_matrix_cpu_RE[j][i] << " ";
    }
    std::cout << std::endl;
  }
  std::cout << std::endl;

  std::cout << "CPU_IM:" << std::endl;
  for(int j=0; j<16; j++){
    for(int i=0; i<16; i++){
      std::cout << dft_matrix_cpu_IM[j][i] << " ";
    }
    std::cout << std::endl;
  }
  std::cout << std::endl;

  std::cout << "GPU_RE:" << std::endl;
  for(int j=0; j<16; j++){
    for(int i=0; i<16; i++){
      std::cout << dft_matrix_gpu_RE[i + 16*j] << " ";
    }
    std::cout << std::endl;
  }
  std::cout << std::endl;

  std::cout << "GPU_IM:" << std::endl;
  for(int j=0; j<16; j++){
    for(int i=0; i<16; i++){
      std::cout << dft_matrix_gpu_IM[i + 16*j] << " ";
    }
    std::cout << std::endl;
  }
  std::cout << std::endl;

  return true;
}
