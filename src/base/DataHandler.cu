#include "hip/hip_runtime.h"
#pragma once

#include <iostream>
#include <optional>
#include <string>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "ComputeDFTMatrix.cu"

//This calss is used to manage the memory on the device needed for the compution
//of one FFT.
//It is intended to be reused if multiple FFTs are to be performed sequentialy.
//Instantiation results in the allocation of the needed memory and the
//precomputation of the DFT matrices that are needed during the computaion.
//The neccesary memcpys to and from the device before and after the computation
//should be performed via the according methods of this class.
//It is recommended to call PeakAtLastError() method after calling the
//constructor to check if the construction was successfull.
class DataHandler{
public:
  DataHandler(int fft_length) : fft_length_(fft_length) {
    if (hipMalloc((void**)(&dptr_data_), 6 * sizeof(__half) * fft_length_)
        != hipSuccess){
       std::cout << hipGetErrorString(hipPeekAtLastError()) << std::endl;
    }
    dptr_input_RE_ = dptr_data_;
    dptr_input_IM_ = dptr_input_RE_ + fft_length_;
    dptr_results_RE_ = dptr_input_IM_ + fft_length_;
    dptr_results_IM_ = dptr_results_RE_ + fft_length_;
    dptr_dft_matrix_RE_ = dptr_results_IM_ + fft_length_;
    dptr_dft_matrix_IM_ = dptr_dft_matrix_RE_ + fft_length_;

    //Here we precompute the dft matrix batches needed for the DFTKernel() and
    //Radix16Kernel(). Currently there is one matrix precomputed for each warp.
    //The other options are to only precompute one (lower memory usage but read
    //conflicts for each warp) and to compute the dft matrix each time during the
    //kernels. (TODO: find out whats "best")
    ComputeDFTMatrix<<<fft_length / 256, 16*16>>>(dptr_dft_matrix_RE_,
                                                  dptr_dft_matrix_IM_);
  }

  std::optional<std::string> PeakAtLastError() {
    if (hipPeekAtLastError() != hipSuccess){
      return hipGetErrorString(hipPeekAtLastError());
    }
    return std::nullopt;
  }

  std::optional<std::string> CopyDataHostToDevice(__half* data) {
    if (hipMemcpy(dptr_input_RE_, data, 2 * fft_length_ * sizeof(__half),
                   hipMemcpyHostToDevice)
         != hipSuccess) {
       return hipGetErrorString(hipPeekAtLastError());
    }

    return std::nullopt;
  }

  std::optional<std::string> CopyResultsDeviceToHost(__half* data,
                                                     int amount_of_r16_steps,
                                                     int amount_of_r2_steps) {
    __half* results;
    if (((amount_of_r16_steps + amount_of_r2_steps) % 2) != 0) {
      results = dptr_results_RE_;
    } else {
      results = dptr_input_RE_;
    }
    if (hipMemcpy(data, results, 2 * fft_length_ * sizeof(__half),
                   hipMemcpyDeviceToHost)
         != hipSuccess) {
       return hipGetErrorString(hipPeekAtLastError());
    }

    return std::nullopt;
  }

  ~DataHandler(){
    hipFree(dptr_data_);
  }

  int fft_length_;
  __half* dptr_data_;
  __half* dptr_input_RE_;
  __half* dptr_input_IM_;
  __half* dptr_results_RE_;
  __half* dptr_results_IM_;
  __half* dptr_dft_matrix_RE_;
  __half* dptr_dft_matrix_IM_;
};

//Similar to the DataHandler class but is used for the async fft compution and
//thus holds the data of the entire batch of ffts to be computed.
class DataBatchHandler{
public:
  DataBatchHandler(int fft_length, int amount_of_ffts) :
      fft_length_(fft_length), amount_of_ffts_(amount_of_ffts) {
    if (hipMalloc((void**)(&dptr_data_),
                   amount_of_ffts_ * 6 * sizeof(__half) * fft_length_)
        != hipSuccess){
       std::cout << hipGetErrorString(hipPeekAtLastError()) << std::endl;
    }

    for(int i=0; i<amount_of_ffts_; i++){
      dptr_input_RE_.resize(amount_of_ffts_, nullptr);
      dptr_input_IM_.resize(amount_of_ffts_, nullptr);
      dptr_results_RE_.resize(amount_of_ffts_, nullptr);
      dptr_results_IM_.resize(amount_of_ffts_, nullptr);
      dptr_dft_matrix_RE_.resize(amount_of_ffts_, nullptr);
      dptr_dft_matrix_IM_.resize(amount_of_ffts_, nullptr);
    }
    for(int i=0; i<amount_of_ffts_; i++){
      dptr_input_RE_[i] = dptr_data_ + (2 * i * fft_length_);
      dptr_input_IM_[i] = dptr_input_RE_[i] + fft_length_;
    }
    for(int i=0; i<amount_of_ffts_; i++){
      dptr_results_RE_[i] = dptr_input_IM_[amount_of_ffts_ - 1] +
                            fft_length_ +
                            (2 * i * fft_length_);
      dptr_results_IM_[i] = dptr_results_RE_[i] + fft_length_;
    }
    for(int i=0; i<amount_of_ffts_; i++){
      dptr_dft_matrix_RE_[i] = dptr_results_IM_[amount_of_ffts_ - 1] +
                               fft_length_ +
                               (2 * i * fft_length_);
      dptr_dft_matrix_IM_[i] = dptr_dft_matrix_RE_[i] + fft_length_;
    }

    //Here we precompute the dft matrix batches needed for the DFTKernel() and
    //Radix16Kernel(). Currently there is one matrix precomputed for each warp.
    //The other options are to only precompute one (lower memory usage but read
    //conflicts for each warp) and to compute the dft matrix each time during the
    //kernels. (TODO: find out whats "best")
    //Create a stream for each fft
    std::vector<hipStream_t> streams;
    streams.resize(amount_of_ffts_);
    for(int i=0; i<amount_of_ffts_; i++){
      hipStreamCreate(&(streams[i]));
    }
    for(int i=0; i<amount_of_ffts_; i++){
      ComputeDFTMatrix<<<fft_length / 256, 16*16, 0, streams[i]>>>(
          dptr_dft_matrix_RE_[i], dptr_dft_matrix_IM_[i]);
    }
  }

  std::optional<std::string> PeakAtLastError() {
    if (hipPeekAtLastError() != hipSuccess){
      return hipGetErrorString(hipPeekAtLastError());
    }
    return std::nullopt;
  }

  std::optional<std::string> CopyDataHostToDevice(__half* data) {
    if (hipMemcpy(dptr_input_RE_[0], data,
                   amount_of_ffts_ * 2 * fft_length_ * sizeof(__half),
                   hipMemcpyHostToDevice)
         != hipSuccess) {
       return hipGetErrorString(hipPeekAtLastError());
    }

    return std::nullopt;
  }

  std::optional<std::string> CopyResultsDeviceToHost(__half* data,
                                                     int amount_of_r16_steps,
                                                     int amount_of_r2_steps) {
    __half* results;
    if (((amount_of_r16_steps + amount_of_r2_steps) % 2) == 1) {
      results = dptr_results_RE_[0];
    } else {
      results = dptr_input_RE_[0];
    }
    if (hipMemcpy(data, results,
                   amount_of_ffts_ * 2 * fft_length_ * sizeof(__half),
                   hipMemcpyDeviceToHost)
         != hipSuccess) {
       return hipGetErrorString(hipPeekAtLastError());
    }

    return std::nullopt;
  }

  ~DataBatchHandler(){
    hipFree((void*)dptr_data_);
  }
  int fft_length_;
  int amount_of_ffts_;
  __half* dptr_data_;
  std::vector<__half*> dptr_input_RE_;
  std::vector<__half*> dptr_input_IM_;
  std::vector<__half*> dptr_results_RE_;
  std::vector<__half*> dptr_results_IM_;
  std::vector<__half*> dptr_dft_matrix_RE_;
  std::vector<__half*> dptr_dft_matrix_IM_;
};

//Multi GPU equivalent to the DataHandler class
//Supports only same length on all GPUs. Diff
class DataHandlerMultiGPU{
public:
  DataHandlerMultiGPU(int fft_length, std::vector<int> device_ids) :
      fft_length_(fft_length){
    device_ids_ = device_ids;
    dptr_data_.resize(device_ids_.size(), nullptr);
    dptr_input_RE_.resize(device_ids_.size(), nullptr);
    dptr_input_IM_.resize(device_ids_.size(), nullptr);
    dptr_results_RE_.resize(device_ids_.size(), nullptr);
    dptr_results_IM_.resize(device_ids_.size(), nullptr);
    dptr_dft_matrix_RE_.resize(device_ids_.size(), nullptr);
    dptr_dft_matrix_IM_.resize(device_ids_.size(), nullptr);

    for(int i=0; i<static_cast<int>(device_ids_.size()); i++){
      hipSetDevice(device_ids_[i]);

      if (hipMalloc((void**)(&dptr_data_[i]), 6 * sizeof(__half) * fft_length_)
          != hipSuccess){
         std::cout << hipGetErrorString(hipPeekAtLastError()) << std::endl;
      }
      dptr_input_RE_[i] = dptr_data_[i];
      dptr_input_IM_[i] = dptr_input_RE_[i] + fft_length_;
      dptr_results_RE_[i] = dptr_input_IM_[i] + fft_length_;
      dptr_results_IM_[i] = dptr_results_RE_[i] + fft_length_;
      dptr_dft_matrix_RE_[i] = dptr_results_IM_[i] + fft_length_;
      dptr_dft_matrix_IM_[i] = dptr_dft_matrix_RE_[i] + fft_length_;

      //Here we precompute the dft matrix batches needed for the DFTKernel() and
      //Radix16Kernel(). Currently there is one matrix precomputed for each warp.
      //The other options are to only precompute one (lower memory usage but read
      //conflicts for each warp) and to compute the dft matrix each time during the
      //kernels. (TODO: find out whats "best")
      ComputeDFTMatrix<<<fft_length / 256, 16*16>>>(dptr_dft_matrix_RE_[i],
                                                    dptr_dft_matrix_IM_[i]);
    }
  }

  std::optional<std::string> PeakAtLastError() {
    for(int i=0; i<static_cast<int>(device_ids_.size()); i++){
      hipSetDevice(device_ids_[i]);
      if (hipPeekAtLastError() != hipSuccess){
        return hipGetErrorString(hipPeekAtLastError());
      }
    }

    return std::nullopt;
  }

  //Copy data to device with id device_ids_[device_id]
  std::optional<std::string> CopyDataHostToDevice(__half* data, int device_id) {
    hipSetDevice(device_ids_[device_id]);
    if (hipMemcpy(dptr_input_RE_[device_id], data,
                   2 * fft_length_ * sizeof(__half), hipMemcpyHostToDevice)
         != hipSuccess) {
       return hipGetErrorString(hipPeekAtLastError());
    }

    return std::nullopt;
  }

  std::optional<std::string> CopyResultsDeviceToHost(__half* data,
                                                     int amount_of_r16_steps,
                                                     int amount_of_r2_steps,
                                                     int device_id) {
    hipSetDevice(device_ids_[device_id]);

    __half* results;
    if (((amount_of_r16_steps + amount_of_r2_steps) % 2) != 0) {
      results = dptr_results_RE_[device_id];
    } else {
      results = dptr_input_RE_[device_id];
    }

    if (hipMemcpy(data, results, 2 * fft_length_ * sizeof(__half),
                   hipMemcpyDeviceToHost)
         != hipSuccess) {
       return hipGetErrorString(hipPeekAtLastError());
    }

    return std::nullopt;
  }

  ~DataHandlerMultiGPU(){
    for(int i=0; i<static_cast<int>(device_ids_.size()); i++){
      hipSetDevice(device_ids_[i]);
      hipFree(dptr_data_[i]);
    }
  }

  int fft_length_;
  std::vector<int> device_ids_;
  std::vector<__half*> dptr_data_;
  std::vector<__half*> dptr_input_RE_;
  std::vector<__half*> dptr_input_IM_;
  std::vector<__half*> dptr_results_RE_;
  std::vector<__half*> dptr_results_IM_;
  std::vector<__half*> dptr_dft_matrix_RE_;
  std::vector<__half*> dptr_dft_matrix_IM_;
};

class DataBatchHandlerMultiGPU{
public:
  DataBatchHandlerMultiGPU(int fft_length, int amount_of_ffts,
                           std::vector<int> device_ids) :
      fft_length_(fft_length), amount_of_ffts_(amount_of_ffts) {
    device_ids_ = device_ids;
    dptr_data_.resize(device_ids_.size());
    dptr_input_RE_.resize(device_ids_.size());
    dptr_input_IM_.resize(device_ids_.size());
    dptr_results_RE_.resize(device_ids_.size());
    dptr_results_IM_.resize(device_ids_.size());
    dptr_dft_matrix_RE_.resize(device_ids_.size());
    dptr_dft_matrix_IM_.resize(device_ids_.size());
    for(int i=0; i<static_cast<int>(device_ids_.size()); i++){
      dptr_input_RE_[i].resize(amount_of_ffts_, nullptr);
      dptr_input_IM_[i].resize(amount_of_ffts_, nullptr);
      dptr_results_RE_[i].resize(amount_of_ffts_, nullptr);
      dptr_results_IM_[i].resize(amount_of_ffts_, nullptr);
      dptr_dft_matrix_RE_[i].resize(amount_of_ffts_, nullptr);
      dptr_dft_matrix_IM_[i].resize(amount_of_ffts_, nullptr);
    }

    for(int j=0; j<static_cast<int>(device_ids_.size()); j++){
      hipSetDevice(device_ids_[j]);

      if (hipMalloc((void**)(&dptr_data_[j]),
                     amount_of_ffts_ * 6 * sizeof(__half) * fft_length_)
          != hipSuccess){
         std::cout << hipGetErrorString(hipPeekAtLastError()) << std::endl;
      }

      for(int i=0; i<amount_of_ffts_; i++){
        dptr_input_RE_[j][i] = dptr_data_[j] + (2 * i * fft_length_);
        dptr_input_IM_[j][i] = dptr_input_RE_[j][i] + fft_length_;
      }
      for(int i=0; i<amount_of_ffts_; i++){
        dptr_results_RE_[j][i] = dptr_input_IM_[j][amount_of_ffts_ - 1] +
                              fft_length_ +
                              (2 * i * fft_length_);
        dptr_results_IM_[j][i] = dptr_results_RE_[j][i] + fft_length_;
      }
      for(int i=0; i<amount_of_ffts_; i++){
        dptr_dft_matrix_RE_[j][i] = dptr_results_IM_[j][amount_of_ffts_ - 1] +
                                 fft_length_ +
                                 (2 * i * fft_length_);
        dptr_dft_matrix_IM_[j][i] = dptr_dft_matrix_RE_[j][i] + fft_length_;
      }

      //Here we precompute the dft matrix batches needed for the DFTKernel() and
      //Radix16Kernel(). Currently there is one matrix precomputed for each warp.
      //The other options are to only precompute one (lower memory usage but read
      //conflicts for each warp) and to compute the dft matrix each time during the
      //kernels. (TODO: find out whats "best")
      //Create a stream for each fft
      std::vector<hipStream_t> streams;
      streams.resize(amount_of_ffts_);
      for(int i=0; i<amount_of_ffts_; i++){
        hipStreamCreate(&(streams[i]));
      }
      for(int i=0; i<amount_of_ffts_; i++){
        ComputeDFTMatrix<<<fft_length / 256, 16*16, 0, streams[i]>>>(
            dptr_dft_matrix_RE_[j][i], dptr_dft_matrix_IM_[j][i]);
      }
    }

  }

  std::optional<std::string> PeakAtLastError() {
    for(int i=0; i<static_cast<int>(device_ids_.size()); i++){
      hipSetDevice(device_ids_[i]);
      if (hipPeekAtLastError() != hipSuccess){
        return hipGetErrorString(hipPeekAtLastError());
      }
    }

    return std::nullopt;
  }

  //Copy data to device with id device_ids_[device_id]
  std::optional<std::string> CopyDataHostToDevice(__half* data, int device_id){
    hipSetDevice(device_ids_[device_id]);
    if (hipMemcpy(dptr_input_RE_[device_id][0], data,
                   amount_of_ffts_ * 2 * fft_length_ * sizeof(__half),
                   hipMemcpyHostToDevice)
         != hipSuccess) {
       return hipGetErrorString(hipPeekAtLastError());
    }

    return std::nullopt;
  }

  std::optional<std::string> CopyResultsDeviceToHost(__half* data,
                                                     int amount_of_r16_steps,
                                                     int amount_of_r2_steps,
                                                     int device_id) {
    hipSetDevice(device_ids_[device_id]);

    __half* results;
    if (((amount_of_r16_steps + amount_of_r2_steps) % 2) == 1) {
      results = dptr_results_RE_[device_id][0];
    } else {
      results = dptr_input_RE_[device_id][0];
    }

    if (hipMemcpy(data, results,
                   amount_of_ffts_ * 2 * fft_length_ * sizeof(__half),
                   hipMemcpyDeviceToHost)
         != hipSuccess) {
       return hipGetErrorString(hipPeekAtLastError());
    }

    return std::nullopt;
  }

  ~DataBatchHandlerMultiGPU(){
    for(int i=0; i<static_cast<int>(device_ids_.size()); i++){
      hipSetDevice(device_ids_[i]);
      hipFree((void*)dptr_data_[i]);
    }
  }

  int fft_length_;
  int amount_of_ffts_;
  std::vector<int> device_ids_;
  std::vector<__half*> dptr_data_;
  std::vector<std::vector<__half*>> dptr_input_RE_;
  std::vector<std::vector<__half*>> dptr_input_IM_;
  std::vector<std::vector<__half*>> dptr_results_RE_;
  std::vector<std::vector<__half*>> dptr_results_IM_;
  std::vector<std::vector<__half*>> dptr_dft_matrix_RE_;
  std::vector<std::vector<__half*>> dptr_dft_matrix_IM_;
};
