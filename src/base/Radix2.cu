//Contains the kernel that performs the radix2 steps
#pragma once

#include <type_traits>

#include <hip/hip_runtime.h>

#include <hip/hip_fp16.h>

//This kernel performs the radix 2 combination steps if neccessary. Since it
//can not utilize tensor cores it is much slower than the radix 16 kernel and
//is only used to allow the compatibility with all input sizes that are powers
//of 2.
//Each thread computes two complex points of the resulting FFT and thus the
//toatl number of threads lauched has to equal sub_fft_length i.e. N/2.
//This kernel performs one combination of 2 N/2 sized ffts and thus if there are
//multiple of those needed for one radix step, multiple kernels have to be
//launched and the ptrs to the in/out data have to point to the beginnning of
//the fft that is to be proccessed and not to the global start of the data.
template <typename Integer>
__global__ void Radix2Kernel(__half* input_data_RE, __half* input_data_IM,
                             __half* output_data_RE, __half* output_data_IM,
                             Integer sub_fft_length) {
  Integer memory_point1_offset = blockDim.x * blockIdx.x + threadIdx.x;
  Integer memory_point2_offset = memory_point1_offset + sub_fft_length;

  //The twiddle factor for the first point is 1 -> only the second point has to
  //be modified
  __half phase =
      __hdiv(__hmul(static_cast<__half>(-M_PI),
                    static_cast<__half>(memory_point1_offset)), sub_fft_length);
  //Modulo version for higher accuracy
  /*
  __half phase =
      __hdiv(__hmul(static_cast<__half>(memory_point1_offset %
                                        (sub_fft_length * 2)),
                    static_cast<__half>(-M_PI)),
             static_cast<__half>(8.0));
  */
  __half twiddle_RE = hcos(phase);
  __half twiddle_IM = hsin(phase);

  //Fetch current data once from global memory to use it twice
  __half input_RE = input_data_RE[memory_point2_offset];
  __half input_IM = input_data_IM[memory_point2_offset];

  //Multiply point 2 with twiddle factor
  __half modified_point2_RE =  input_RE * twiddle_RE - input_IM * twiddle_IM;
  __half modified_point2_IM =  input_RE * twiddle_IM + input_IM * twiddle_RE;

  //Combine FFTs
  output_data_RE[memory_point1_offset] =
      input_data_RE[memory_point1_offset] + modified_point2_RE;
  output_data_IM[memory_point1_offset] =
      input_data_IM[memory_point1_offset] + modified_point2_IM;

  output_data_RE[memory_point2_offset] =
      input_data_RE[memory_point1_offset] - modified_point2_RE;
  output_data_IM[memory_point2_offset] =
      input_data_IM[memory_point1_offset] - modified_point2_IM;

  //For sequential scaling
  /*
  output_data_RE[memory_point1_offset] =
      __hmul(input_data_RE[memory_point1_offset] + modified_point2_RE,
             static_cast<__half>(0.5));
  output_data_IM[memory_point1_offset] =
      __hmul(input_data_IM[memory_point1_offset] + modified_point2_IM,
             static_cast<__half>(0.5));

  output_data_RE[memory_point2_offset] =
      __hmul(input_data_RE[memory_point1_offset] - modified_point2_RE,
             static_cast<__half>(0.5));
  output_data_IM[memory_point2_offset] =
      __hmul(input_data_IM[memory_point1_offset] - modified_point2_IM,
             static_cast<__half>(0.5));
  */
}
