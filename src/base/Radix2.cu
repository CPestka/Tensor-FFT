//Contains the kernel that performs the radix2 steps
#pragma once

#include <type_traits>

#include <hip/hip_runtime.h>

#include <hip/hip_fp16.h>

//This kernel performs the radix 2 combination steps if neccessary. Since it
//can not utilize tensor cores it is much slower than the radix 16 kernel and
//is only used to allow the compatibility with all input sizes that are powers
//of 2.
//Each thread computes two complex points of the resulting FFT and thus the
//toatl number of threads lauched has to equal sub_fft_length i.e. N/2.
//This kernel performs one combination of 2 N/2 sized ffts and thus if there are
//multiple of those needed for one radix step, multiple kernels have to be
//launched and the ptrs to the in/out data have to point to the beginnning of
//the fft that is to be proccessed and not to the global start of the data.
template <typename Integer>
__global__ void Radix2Kernel(__half* input_data_RE, __half* input_data_IM,
                             __half* output_data_RE, __half* output_data_IM,
                             Integer sub_fft_length) {
  Integer memory_point1_offset = blockDim.x * blockIdx.x + threadIdx.x;
  Integer memory_point2_offset = memory_point1_offset + sub_fft_length;

  //The twiddle factor for the first point is 1 -> only the second point has to
  //be modified
  //Compute phase = -2PI*i/fft_length
  //Use float to prevent overflow of large ints memory_point1_offset and
  //sub_fft_length
  float tmp = static_cast<float>(memory_point1_offset) /
              static_cast<float>(sub_fft_length);
  __half phase = __hmul(static_cast<__half>(M_PI), static_cast<__half>(tmp));

  __half twiddle_RE = hcos(phase);
  __half twiddle_IM = -hsin(phase);

  //Fetch current data once from global memory to use it twice
  __half point2_RE = input_data_RE[memory_point2_offset];
  __half point2_IM = input_data_IM[memory_point2_offset];

  //Multiply point 2 with twiddle factor
  __half modified_point2_RE =
      __hsub(__hmul(point2_RE, twiddle_RE), __hmul(point2_IM, twiddle_IM));
  __half modified_point2_IM =
      __hfma(point2_RE , twiddle_IM, __hmul(point2_IM, twiddle_RE));

  //Load point 1 from global mem once to use it twice
  __half point1_RE = input_data_RE[memory_point1_offset];
  __half point1_IM = input_data_IM[memory_point1_offset];

  //Combine FFTs

  //For unscaled or scaling at once
  // output_data_RE[memory_point1_offset] =
  //     __hadd(point1_RE, modified_point2_RE);
  // output_data_IM[memory_point1_offset] =
  //     __hadd(point1_IM, modified_point2_IM);
  //
  // output_data_RE[memory_point2_offset] =
  //     __hadd(point1_RE, modified_point2_RE);
  // output_data_IM[memory_point2_offset] =
  //     __hadd(point1_IM, modified_point2_IM);

  //For sequential scaling
  output_data_RE[memory_point1_offset] =
      __hmul(__hadd(point1_RE, modified_point2_RE), static_cast<__half>(0.5));
  output_data_IM[memory_point1_offset] =
      __hmul(__hadd(point1_IM, modified_point2_IM), static_cast<__half>(0.5));

  output_data_RE[memory_point2_offset] =
      __hmul(__hsub(point1_RE, modified_point2_RE), static_cast<__half>(0.5));
  output_data_IM[memory_point2_offset] =
      __hmul(__hsub(point1_IM, modified_point2_IM), static_cast<__half>(0.5));

  printf("ID: %d tmp: %f phase: %f twid_RE: %f twid_IM %f p1_RE: %f p1_IM: %f p2_RE: %f p2_IM: %f p2Mod_RE: %f p2Mod_IM: %f p1Out_RE: %f p1Out_IM: %f p2Out_RE: %f p2Out_IM: %f", memory_point1_offset, tmp, static_cast<float>(phase), static_cast<float>(twiddle_RE), static_cast<float>(twiddle_IM), static_cast<float>(point1_RE), static_cast<float>(point1_IM), static_cast<float>(point2_RE), static_cast<float>(point2_IM), static_cast<float>(modified_point2_RE), static_cast<float>(modified_point2_IM), static_cast<float>(output_data_RE[memory_point1_offset]), static_cast<float>(output_data_IM[memory_point1_offset]), static_cast<float>(output_data_RE[memory_point2_offset]), static_cast<float>(output_data_RE[memory_point2_offset]));
}
