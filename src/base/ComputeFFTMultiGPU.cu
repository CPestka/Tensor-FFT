#include "hip/hip_runtime.h"
//Multi GPU version of the function ComputeFFTs()
#pragma once
#include <vector>
#include <optional>
#include <string>
#include <thread>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "ComputeFFT.cu"
#include "Plan.cpp"
#include "DataHandler.cu"
#include "../testing/FileWriter.cu"

void SingleGPUWork(int device_id, std::vector<Plan> &fft_plans,
                   std::vector<DataHandler> &data,
                   std::vector<hipStream_t> &streams){
  hipSetDevice(device_id);

  std::optional<std::string> single_GPU_error = ComputeFFTs(fft_plans, data,
                                                            streams);

  if (single_GPU_error) {
    WriteLogToFile(("Device" + std::to_string(device_id)) + "Error.log",
                   single_GPU_error.value());
  }
}

//Simple implementation for multiple GPUs of ComputeFFTs(). device_list[i]
//contains the id of the GPU to be used for the call of
//ComputeFFTs(fft_plans[i], data[i]). The ids HAVE to be unique and correspond
//to an existing device id of the used system.
//Errors are writen to log files.
//This function is not blocking in respect to the CPU i.e. it likely returns
//before the work on the devices is finished.
void ComputeFFTsMultiGPU(std::vector<int> device_list,
                         std::vector<std::vector<Plan>> &fft_plans,
                         std::vector<std::vector<DataHandler>> &data,
                         std::vector<std::vector<hipStream_t>> &streams){
  std::vector<std::thread> worker;
  for(int i=0; i<static_cast<int>(device_list.size()); i++){
    worker.push_back(std::thread(&SingleGPUWork, device_list[i], fft_plans[i],
                                 data[i], streams[i]));
  }

  for(int i=0; i<static_cast<int>(device_list.size()); i++){
    worker[i].join();
  }
}
