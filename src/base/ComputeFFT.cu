#include "hip/hip_runtime.h"
//Contains Functions to compute ffts of half precission data. The neccesary
//cpying operations to and from the GPU are handled by methods of the
//DataHandler class which also holds the device ptr to the according data.
//The parameters of the fft are hold in the struct Plan, which should be
//produced via the function CreatePlan().
//The computation n ffts of a one given length is typicaly performed the
//following way: 1. Create Plan 2. Create DataHandler 3.1. Cpy data to GPU using
//e.g. CopyDataHostToDevice() method of DataHandler 3.2. Call function
//ComputeFFT() 3.3 Cpy results back via method CopyResultsDeviceToHost()
//Repeating step 3. n times.
//Due to the usage of tensor cores the minimal input size is 16^2. All
//other powers of of two are supported as input sizes. Performance is expected
//to be best (compared to other fft libaries) if the input size N is large and
//if N= 16^l * 2^k (while keeping k as small as possible) k is small. This is
//due to the fact that the radix 16 part of the algorithm is accelerated by
//tensor cores compared to the radix 2 part for which this is not the case.
#pragma once

#include <iostream>
#include <vector>
#include <optional>
#include <string>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "Transposer.cu"
#include "TensorDFT16.cu"
#include "TensorRadix16.cu"
#include "Radix2.cu"
#include "Plan.cpp"
#include "DataHandler.cu"

//Computes a sigle FFT.
//If the GPU isnt satureted with one FFT and there are multiple FFTs to compute
//using the async version below should increase performance.
std::optional<std::string> ComputeFFT(Plan &fft_plan, DataHandler &data){
  //Launch kernel that performs the transposes to prepare the data for the
  //radix steps
  TransposeKernel<<<fft_plan.transposer_amount_of_blocks_,
                    fft_plan.transposer_blocksize_>>>(
      data.dptr_input_RE_, data.dptr_input_IM_, data.dptr_results_RE_,
      data.dptr_results_IM_, fft_plan.fft_length_,
      fft_plan.amount_of_r16_steps_, fft_plan.amount_of_r2_steps_);

  //Launch baselayer DFT step kernel
  DFTKernel<<<fft_plan.dft_amount_of_blocks_,
              32 * fft_plan.dft_warps_per_block_>>>(
      data.dptr_results_RE_, data.dptr_results_IM_, data.dptr_input_RE_,
      data.dptr_input_IM_, data.dptr_dft_matrix_RE_, data.dptr_dft_matrix_IM_);

  __half* dptr_current_input_RE;
  __half* dptr_current_input_IM;
  __half* dptr_current_results_RE;
  __half* dptr_current_results_IM;
  int sub_fft_length = 16;

  //Launch radix16 kernels
  for(int i=0; i<fft_plan.amount_of_r16_steps_; i++){
    //For each step the input data is the output data of the previous step
    if ((i % 2) == 0) {
      dptr_current_input_RE = data.dptr_input_RE_;
      dptr_current_input_IM = data.dptr_input_IM_;
      dptr_current_results_RE = data.dptr_results_RE_;
      dptr_current_results_IM = data.dptr_results_IM_;
    } else {
      dptr_current_input_RE = data.dptr_results_RE_;
      dptr_current_input_IM = data.dptr_results_IM_;
      dptr_current_results_RE = data.dptr_input_RE_;
      dptr_current_results_IM = data.dptr_input_IM_;
    }

    int shared_mem_in_bytes = fft_plan.r16_warps_per_block_ * 16 * 16 *
                              2 * sizeof(__half);

    if (i == 0) {
      hipFuncSetAttribute(reinterpret_cast<const void*>(Radix16KernelFirstStep),
                           hipFuncAttributeMaxDynamicSharedMemorySize,
                           shared_mem_in_bytes);
      Radix16KernelFirstStep<<<fft_plan.r16_amount_of_blocks_,
                               32 * fft_plan.r16_warps_per_block_,
                               shared_mem_in_bytes>>>(
          dptr_current_input_RE, dptr_current_input_IM,
          dptr_current_results_RE, dptr_current_results_IM,
          data.dptr_dft_matrix_RE_, data.dptr_dft_matrix_IM_);
    } else {
      hipFuncSetAttribute(reinterpret_cast<const void*>(Radix16Kernel),
                           hipFuncAttributeMaxDynamicSharedMemorySize,
                           shared_mem_in_bytes);
      Radix16Kernel<<<fft_plan.r16_amount_of_blocks_,
                      32 * fft_plan.r16_warps_per_block_,
                      shared_mem_in_bytes>>>(
          dptr_current_input_RE, dptr_current_input_IM, dptr_current_results_RE,
          dptr_current_results_IM, data.dptr_dft_matrix_RE_,
          data.dptr_dft_matrix_IM_, fft_plan.fft_length_, sub_fft_length, i);
    }

    //Update sub_fft_length
    sub_fft_length = sub_fft_length * 16;
  }

  //Radix 2 kernels
  for(int i=0; i<fft_plan.amount_of_r2_steps_; i++){
    //For each step the input data is the output data of the previous step
    if (((i + fft_plan.amount_of_r16_steps_) % 2) == 0) {
      dptr_current_input_RE = data.dptr_results_RE_;
      dptr_current_input_IM = data.dptr_results_IM_;
      dptr_current_results_RE = data.dptr_input_RE_;
      dptr_current_results_IM = data.dptr_input_IM_;
    } else {
      dptr_current_input_RE = data.dptr_input_RE_;
      dptr_current_input_IM = data.dptr_input_IM_;
      dptr_current_results_RE = data.dptr_results_RE_;
      dptr_current_results_IM = data.dptr_results_IM_;
    }

    int remaining_sub_ffts = 1;
    for(int k=0; k<fft_plan.amount_of_r2_steps_ - i; k++){
      remaining_sub_ffts = remaining_sub_ffts * 2;
    }

    int amount_of_r2_blocks = sub_fft_length / fft_plan.r2_blocksize_;

    //One radix2 kernel combines 2 subffts -> if there are still more than 2
    //launch multiple kernels
    for(int j=0; j<(remaining_sub_ffts/2); j++){
      int memory_offset = j * 2 * sub_fft_length;
      Radix2Kernel<<<amount_of_r2_blocks, fft_plan.r2_blocksize_>>>(
          dptr_current_input_RE + memory_offset,
          dptr_current_input_IM + memory_offset,
          dptr_current_results_RE + memory_offset,
          dptr_current_results_IM + memory_offset,
          sub_fft_length);
    }

    //Update sub_fft_length
    sub_fft_length = sub_fft_length * 2;
  }

  return std::nullopt;
}

//Similar to ComputeFFT() but accepts multiple ffts at a time. For each fft
//respectively the corresponding memcpys and kernels are issued into one stream
//respectively, which allows work for multiple ffts to be executed concurrently
//if the recources on the device are avaiable.
//The memory requiredments are the same as for ComputeFFT() but added together
//for each fft.
std::optional<std::string> ComputeFFTs(Plan &fft_plans,
                                       DataBatchHandler &data,
                                       std::vector<hipStream_t> &streams){
  //Launch kernel that performs the transposes to prepare the data for the
  //radix steps
  for(int i=0; i<data.amount_of_ffts_; i++){
    TransposeKernel<<<fft_plans.transposer_amount_of_blocks_,
                      fft_plans.transposer_blocksize_, 0, streams[i]>>>(
        data.dptr_input_RE_[i], data.dptr_input_IM_[i],
        data.dptr_results_RE_[i], data.dptr_results_IM_[i],
        fft_plans.fft_length_, fft_plans.amount_of_r16_steps_,
        fft_plans.amount_of_r2_steps_);
  }

  //Launch baselayer DFT step kernel
  for(int i=0; i<data.amount_of_ffts_; i++){
    DFTKernel<<<fft_plans.dft_amount_of_blocks_,
                32 * fft_plans.dft_warps_per_block_, 0, streams[i]>>>(
        data.dptr_results_RE_[i], data.dptr_results_IM_[i],
        data.dptr_input_RE_[i], data.dptr_input_IM_[i],
        data.dptr_dft_matrix_RE_[i], data.dptr_dft_matrix_IM_[i]);
  }

  std::vector<__half*> dptr_current_input_RE;
  std::vector<__half*> dptr_current_input_IM;
  std::vector<__half*> dptr_current_results_RE;
  std::vector<__half*> dptr_current_results_IM;
  dptr_current_input_RE.resize(data.amount_of_ffts_, nullptr);
  dptr_current_input_IM.resize(data.amount_of_ffts_, nullptr);
  dptr_current_results_RE.resize(data.amount_of_ffts_, nullptr);
  dptr_current_results_IM.resize(data.amount_of_ffts_, nullptr);

  int sub_fft_length = 16;

  int shared_mem_in_bytes = fft_plans.r16_warps_per_block_ *
                            16 * 16 * 2 * sizeof(__half);
  hipFuncSetAttribute(reinterpret_cast<const void*>(Radix16KernelFirstStep),
                       hipFuncAttributeMaxDynamicSharedMemorySize,
                       shared_mem_in_bytes);
  hipFuncSetAttribute(reinterpret_cast<const void*>(Radix16Kernel),
                       hipFuncAttributeMaxDynamicSharedMemorySize,
                       shared_mem_in_bytes);

  for(int i=0; i<data.amount_of_ffts_; i++){
    for(int j=0; j<fft_plans.amount_of_r16_steps_; j++){
      //For each step the input data is the output data of the previous step
      if ((j % 2) == 0) {
        dptr_current_input_RE[i] = data.dptr_input_RE_[i];
        dptr_current_input_IM[i] = data.dptr_input_IM_[i];
        dptr_current_results_RE[i] = data.dptr_results_RE_[i];
        dptr_current_results_IM[i] = data.dptr_results_IM_[i];
      } else {
        dptr_current_input_RE[i] = data.dptr_results_RE_[i];
        dptr_current_input_IM[i] = data.dptr_results_IM_[i];
        dptr_current_results_RE[i] = data.dptr_input_RE_[i];
        dptr_current_results_IM[i] = data.dptr_input_IM_[i];
      }

      if (j == 0) {
        Radix16KernelFirstStep<<<fft_plans.r16_amount_of_blocks_,
                                 32 * fft_plans.r16_warps_per_block_,
                                 shared_mem_in_bytes, streams[i]>>>(
            dptr_current_input_RE[i], dptr_current_input_IM[i],
            dptr_current_results_RE[i], dptr_current_results_IM[i],
            data.dptr_dft_matrix_RE_[i], data.dptr_dft_matrix_IM_[i]);
      } else {
        Radix16Kernel<<<fft_plans.r16_amount_of_blocks_,
                       32 * fft_plans.r16_warps_per_block_,
                       shared_mem_in_bytes, streams[i]>>>(
            dptr_current_input_RE[i], dptr_current_input_IM[i],
            dptr_current_results_RE[i], dptr_current_results_IM[i],
            data.dptr_dft_matrix_RE_[i], data.dptr_dft_matrix_IM_[i],
            fft_plans.fft_length_, sub_fft_length, j);
      }

      //Update sub_fft_length
      sub_fft_length = sub_fft_length * 16;
    }
  }

  for(int i=0; i<data.amount_of_ffts_; i++){
    //Radix 2 kernels
    for(int j=0; j<fft_plans.amount_of_r2_steps_; j++){
      //For each step the input data is the output data of the previous step
      if (((j + fft_plans.amount_of_r16_steps_) % 2) == 0) {
        dptr_current_input_RE[i] = data.dptr_input_RE_[i];
        dptr_current_input_IM[i] = data.dptr_input_IM_[i];
        dptr_current_results_RE[i] = data.dptr_results_RE_[i];
        dptr_current_results_IM[i] = data.dptr_results_IM_[i];
      } else {
        dptr_current_input_RE[i] = data.dptr_results_RE_[i];
        dptr_current_input_IM[i] = data.dptr_results_IM_[i];
        dptr_current_results_RE[i] = data.dptr_input_RE_[i];
        dptr_current_results_IM[i] = data.dptr_input_IM_[i];
      }

      int amount_of_r2_blocks = sub_fft_length / fft_plans.r2_blocksize_;

      int remaining_sub_ffts = 1;
      for(int k=0; k<fft_plans.amount_of_r2_steps_ - j; k++){
        remaining_sub_ffts = remaining_sub_ffts * 2;
      }

      //One radix2 kernel combines 2 subffts -> if there are still more than 2
      //launch multiple kernels
      for(int k=0; k<(remaining_sub_ffts/2); k++){
        int memory_offset = k * sub_fft_length;
        Radix2Kernel<<<amount_of_r2_blocks, fft_plans.r2_blocksize_,
                       0, streams[i]>>>(
            dptr_current_input_RE[i] + memory_offset,
            dptr_current_input_IM[i] + memory_offset,
            dptr_current_results_RE[i] + memory_offset,
            dptr_current_results_IM [i]+ memory_offset,
            sub_fft_length);
      }

      //Update sub_fft_length
      sub_fft_length = sub_fft_length * 2;
    }
  }

  return std::nullopt;
}
