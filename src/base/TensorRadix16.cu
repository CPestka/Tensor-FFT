//Contains the kernel that performs the radix16 steps on tensor cores
#pragma once

#include <hip/hip_runtime.h>

#include <hip/hip_fp16.h>
#include <mma.h>
using namespace nvcuda;

//These kernels performs the radix 16 recombination steps. It takes 16 length
//N/(16^m) with m being the number of already performed radix 16 steps + 1 and
//combines them to a 16^(m+1) length FFT. Multiple of these results are then
//used as the input for another call of this kernel or the Radix2Kernel() untill
//the final FFT length is reached.
//The Kernel can be divided into 3 main sections. Contrary to the DFTKernel()
//the input data cant just be read and used directly by the tensor cores.
//Instead a componentwise multiplication with the so called twiddle factors has
//to be performed first. Due to this in the first section each warp loads the
//input data, computes the multiplication and stores the result in its own
//section of a shared memory buffer.
//In the second and third section the data is then loaded into the fragments
//and the matrix multiplication of the 16^mx16 data matrix with the 16x16 DFT
//matrix is performed. For m > 1 the matrix multiplication is split into m
//16x16 * 16x16 matrixmultiplications and the results are then recombined
//by storing the results in the correct place in memory. Also due to this the
//input data for m > 1 isnt linear in memory but for one 16x16 matrix instead
//we have 16 linear chuncks of length 16 that are each offset to each other by
//sub_fft_length=16^(m+1).

//Variant of the kernel described above that is only used for the first R16
//step, due to tha fact that the calculation can be stated much cheaper for the
//first step.
__global__ void Radix16KernelFirstStep(__half* input_data_RE,
                                       __half* input_data_IM,
                                       __half* output_data_RE,
                                       __half* output_data_IM,
                                       __half* dft_matrix_batch_RE,
                                       __half* dft_matrix_batch_IM) {
  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
  int warp_id = thread_id / 32;
  int inter_warp_id = thread_id % 32;
  int inter_block_warp_id = warp_id % (blockDim.x / 32);
  //Used to devide work for threads in a warp since the problem size is 16 based
  //and the tensor core operations are "warp wide".
  int inter_warp_id_16 = inter_warp_id % 16;
  int inter_warp_id_is_upper_16 = inter_warp_id / 16;

  //Declare the fragments
  //The needed matrix multiplication is normaly data x dft, but since the data
  //in global memory is stored in row major but would be needed here in collum
  //major order, we instead compute: (data x dft)^T = dft^T x data^T =
  //dft x data^T because dft is symetrical.
  wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major>
      dft_RE_frag;
  wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major>
      dft_IM_frag;
  wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major>
      data_RE_frag;
  wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major>
      data_IM_frag;
  wmma::fragment<wmma::accumulator, 16, 16, 16, half> accumulator_RE_1_frag;
  wmma::fragment<wmma::accumulator, 16, 16, 16, half> accumulator_RE_2_frag;
  wmma::fragment<wmma::accumulator, 16, 16, 16, half> accumulator_IM_frag;

  //Initialize the output to zero
  wmma::fill_fragment(accumulator_RE_1_frag, 0.0f);
  wmma::fill_fragment(accumulator_RE_2_frag, 0.0f);
  wmma::fill_fragment(accumulator_IM_frag, 0.0f);

  //Load the inputs
  int warp_memory_offset = 256 * warp_id;
  wmma::load_matrix_sync(dft_RE_frag, dft_matrix_batch_RE + warp_memory_offset,
                         16);
  wmma::load_matrix_sync(dft_IM_frag, dft_matrix_batch_IM + warp_memory_offset,
                         16);

  //Since fragments can only be accessed uniformly multiplication with different
  //factors to each element isnt possible.
  //Due to this we utilize a shared memory buffer of size of the data
  //for this block -> amount_of_warps_per_block * size_of_fragment (16*16) *
  //2 (RE + IM) * sizeof(half); (blockdim.x / 32) = amount_of_warps_per_block
  //For recomended amount_of_warps_per_block=4 -> 4kB
  extern __shared__ __half buffer[];
  __half* buffer_RE = buffer + (512 * inter_block_warp_id);
  __half* buffer_IM = buffer_RE + 256;

  //In this case one warp performs a combination of 16 size 16 FFTs. This means
  //that the resulting data does not need to be rearanged.
  //Each of the 32 threads per warp loads 8 (8*32=16*16) data points, multiplies
  //them with the twiddle factors and stores the now prepared data in the
  //fragment.
  #pragma unroll
  for(int k=0; k<8; k++){
    int j = k + (8 * inter_warp_id_is_upper_16);
    int memory_offset = (inter_warp_id_16 + 16*j);
    int global_memory_offset = warp_memory_offset + memory_offset;

    //Compute RE and IM of twiddle factors
    float phase = (2 * M_PI * inter_warp_id_16 * j) / 256;
    //TO-SELF: test __cosf vs cos accuracy and speed
    __half twiddle_RE = __float2half(cos(phase));
    __half twiddle_IM = __float2half(-sin(phase));

    //Fetch current data once from global memory to use it twice
    __half input_RE = input_data_RE[global_memory_offset];
    __half input_IM = input_data_IM[global_memory_offset];

    //Store modified data to buffer arrays
    //mod_RE = RE*twid_RE - IM*twid_IM
    buffer_RE[memory_offset] =
        __hsub(__hmul(input_RE, twiddle_RE), __hmul(input_IM, twiddle_IM));

    //mod_IM = RE*twid_IM + IM*twid_RE
    buffer_IM[memory_offset] =
        __hfma(input_RE , twiddle_IM, __hmul(input_IM, twiddle_RE));
  }

  //Load the modified data from shared mem buffer
  wmma::load_matrix_sync(data_RE_frag, buffer_RE, 16);
  wmma::load_matrix_sync(data_IM_frag, buffer_IM, 16);

  //Perform the matrix multiplication of two complex matrices AxB via 4 matrix
  //multiplications i.e. RE(AxB)=RE(A)xRE(B) - IM(A)xIM(B) and IM(AxB) =
  //RE(A)xIM(B) + IM(A)xRE(B)
  wmma::mma_sync(accumulator_RE_1_frag, dft_RE_frag, data_RE_frag,
                 accumulator_RE_1_frag);
  wmma::mma_sync(accumulator_RE_2_frag, dft_IM_frag, data_IM_frag,
                 accumulator_RE_2_frag);
  wmma::mma_sync(accumulator_IM_frag, dft_IM_frag, data_RE_frag,
                 accumulator_IM_frag);
  wmma::mma_sync(accumulator_IM_frag, dft_RE_frag, data_IM_frag,
                 accumulator_IM_frag);

  //Store IM part of the output
  wmma::store_matrix_sync(output_data_IM + warp_memory_offset,
                          accumulator_IM_frag, 16, wmma::mem_row_major);

  #pragma unroll
  for(int i=0; i<accumulator_RE_1_frag.num_elements; i++){
    output_data_RE[warp_memory_offset + i] =
        __hsub(accumulator_RE_1_frag.x[i],
               accumulator_RE_2_frag.x[i]);
  }
}

//Variant of the kernel described above that is only used for the mth step with
//m>0
__global__ void Radix16Kernel(__half* input_data_RE, __half* input_data_IM,
                              __half* output_data_RE, __half* output_data_IM,
                              __half* dft_matrix_batch_RE,
                              __half* dft_matrix_batch_IM,
                              int fft_length, int sub_fft_length,
                              int current_radix16_step) {
  int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
  int warp_id = thread_id / 32;
  int inter_warp_id = thread_id % 32;
  int inter_block_warp_id = warp_id % (blockDim.x / 32);
  //Used to devide work for threads in a warp since the problem size is 16 based
  //and the tensor core operations are "warp wide".
  int inter_warp_id_16 = inter_warp_id % 16;
  int inter_warp_id_is_upper_16 = inter_warp_id / 16;

  //Declare the fragments
  wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major>
      data_RE_frag;
  wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major>
      data_IM_frag;
  wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major>
      dft_RE_frag;
  wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major>
      dft_IM_frag;
  wmma::fragment<wmma::accumulator, 16, 16, 16, half> accumulator_RE_1_frag;
  wmma::fragment<wmma::accumulator, 16, 16, 16, half> accumulator_RE_2_frag;
  wmma::fragment<wmma::accumulator, 16, 16, 16, half> accumulator_IM_frag;

  //Initialize the output to zero
  wmma::fill_fragment(accumulator_RE_1_frag, 0.0f);
  wmma::fill_fragment(accumulator_RE_2_frag, 0.0f);
  wmma::fill_fragment(accumulator_IM_frag, 0.0f);

  //Load the inputs
  int warp_memory_offset = 256 * warp_id;
  wmma::load_matrix_sync(dft_RE_frag, dft_matrix_batch_RE + warp_memory_offset,
                         16);
  wmma::load_matrix_sync(dft_IM_frag, dft_matrix_batch_IM + warp_memory_offset,
                         16);

  //Since fragments can only be accessed uniformly the reordering of the results
  //when storing them back to memory can not be done directly with the fragments
  //at all.
  //Due to this purpose we utilize a shared memory buffer of size of the data
  //for this block -> amount_of_warps_per_block * size_of_fragment (16*16) *
  //2 (RE + IM) * sizeof(half); (blockdim.x / 32) = amount_of_warps_per_block
  //For recomended amount_of_warps_per_block=4 -> 4kB -> ok on A100
  extern __shared__ __half buffer[];
  __half* buffer_RE = buffer + (512 * inter_block_warp_id);
  __half* buffer_IM = buffer_RE + 256;

  int combined_fft_length = sub_fft_length * 16;
  int amount_of_warps_pes_substep = sub_fft_length / 16;
  int inter_substep_id = warp_id % amount_of_warps_pes_substep;
  int substep_id = warp_id / amount_of_warps_pes_substep;

  //Each of the 32 threads pre warp loads 8 (8*32=16*16) data
  //points. However the data of the needed 16x16 matrix of input data is not
  //linaer in memory. The entire 16^mx16 matrix (which is linear in memory) is
  //divided into m 16x16 matrices. This means that the data for one 16x16
  //matrix consists of 16 length 16 linear chuncks, which are offset in
  //respect to each other by sub_fft_length=16^m.
  //Also, by swaping the indecies when loading the storing to and from the
  //fragment the fragment holds the transposed data, which is needed since the
  //data is stored in row major order in memory but is needed in collum major
  //for the matrix multiplication.
  #pragma unroll
  for(int k=0; k<8; k++){
    int i = inter_warp_id_16 + (inter_substep_id * 16);
    int j = k + (8 * inter_warp_id_is_upper_16);
    int global_memory_offset = i +
                               sub_fft_length * j +
                               substep_id * combined_fft_length;
    int buffer_matrix_memory_offset = j + 16 * inter_warp_id_16;

    //Compute twiddle factors
    float phase = (2 * M_PI * i * j) / combined_fft_length;
    //TO-SELF: test __cosf vs cos accuracy and speed
    __half twiddle_RE = __float2half(cos(phase));
    __half twiddle_IM = __float2half(-sin(phase));

    //Fetch current data once from global memory to use it twice
    __half input_RE = input_data_RE[global_memory_offset];
    __half input_IM = input_data_IM[global_memory_offset];

    //Store modified data to buffer arrays
    //mod_RE = RE*twid_RE - IM*twid_IM
    buffer_RE[buffer_matrix_memory_offset] =
        __hsub(__hmul(input_RE, twiddle_RE), __hmul(input_IM, twiddle_IM));
    //mod_IM = RE*twid_IM + IM*twid_RE
    buffer_IM[buffer_matrix_memory_offset] =
        __hfma(input_RE , twiddle_IM, __hmul(input_IM, twiddle_RE));
  }

  //Load the modified data from shared mem buffer
  wmma::load_matrix_sync(data_RE_frag, buffer_RE, 16);
  wmma::load_matrix_sync(data_IM_frag, buffer_IM, 16);

  //Perform the matrix multiplication of two complex matrices AxB via 4 matrix
  //multiplications i.e. RE(AxB)=RE(A)xRE(B) - IM(A)xIM(B) and IM(AxB) =
  //RE(A)xIM(B) + IM(A)xRE(B)
  wmma::mma_sync(accumulator_RE_1_frag, data_RE_frag, dft_RE_frag,
                 accumulator_RE_1_frag);
  wmma::mma_sync(accumulator_RE_2_frag, data_IM_frag, dft_IM_frag,
                 accumulator_RE_2_frag);
  wmma::mma_sync(accumulator_IM_frag, data_RE_frag, dft_IM_frag,
                 accumulator_IM_frag);
  wmma::mma_sync(accumulator_IM_frag, data_IM_frag, dft_RE_frag,
                 accumulator_IM_frag);

  //Store results to buffer
  wmma::store_matrix_sync(buffer_IM, accumulator_IM_frag, 16,
                          wmma::mem_row_major);
  #pragma unroll
  for(int i=0; i<accumulator_RE_1_frag.num_elements; i++){
    buffer_RE[i] = __hsub(accumulator_RE_1_frag.x[i],
                          accumulator_RE_2_frag.x[i]);
  }

  //Store the results in the appropriately reordered way into the output array
  //The data is stored back the way it was intialy the i.e. a 16^mx16 linear=
  //row-major array and is then reinterpreted as a linear in memory FFT of
  //length 16^(m+1)
  //The transpose operation is also reverted.
  #pragma unroll
  for(int k=0; k<8; k++){
    int i = inter_warp_id_16 + (inter_substep_id * 16);
    int j = k + (8 * inter_warp_id_is_upper_16);
    int global_memory_offset = i +
                               sub_fft_length * j +
                               substep_id * combined_fft_length;
    int buffer_matrix_memory_offset = j + 16 * inter_warp_id_16;

    output_data_RE[global_memory_offset] =
        buffer_RE[buffer_matrix_memory_offset];
    output_data_IM[global_memory_offset] =
        buffer_IM[buffer_matrix_memory_offset];
  }
}
